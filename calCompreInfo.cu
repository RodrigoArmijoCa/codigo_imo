#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hipblas.h"
#include <hip/hip_runtime.h>
#include <float.h>
#include <math.h>
#include <sys/time.h>
#include <cblas.h>
#include <f77blas.h>
#include <pthread.h>
#include <assert.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <time.h>
#include <arrayfire.h>
#include <af/hip/hip_runtime.h>
#include <fitsio.h>
#include <cublasXt.h>
#include <gsl/gsl_math.h>
#include <gsl/gsl_vector.h>
#include <gsl/gsl_filter.h>
#include <gsl/gsl_min.h>


// rarmijo@158.170.35.147

//nvcc otroconfloat.cu -lcudart -lcublas -lcuda -lblasx -I/opt/arrayfire/include/ -L/opt/arrayfire/lib64/ -lafcuda -lcfitsio -o otroconfloat
/* sudo scp /home/yoyisaurio/Desktop/juguetes\ de\ CUDA/otroconfloat.cu rarmijo@158.170.35.139:/home/rarmijo/Desktop/ */


// nvcc calCompreInfo.cu -lcudart -lcublas -lcuda -lblasx -I/opt/arrayfire/include/ -L/opt/arrayfire/lib64/ -lafcuda -lcfitsio -lgsl -lgslcblas -lm -o calCompreInfo
// sudo scp /home/yoyisaurio/Desktop/proyecto/calCompreInfo.cu rarmijo@beam.diinf.usach.cl:/home/rarmijo
// nvcc calCompreInfo.cu -lcudart -lcublas -lcuda -lblasx -I/opt/arrayfire/include/ -L/opt/arrayfire/lib64/ -lafcuda -lcfitsio -lgsl -lgslcblas -lm -o calCompreInfo
// ./calCompreInfo

// sudo scp /home/yoyisaurio/Desktop/proyecto/nuevo.cu rarmijo@158.170.35.139:/home/rarmijo/Desktop/proyecto

// sudo scp rarmijo@beam.diinf.usach.cl:/home/rarmijo/float_calCompresion_baseNormal_cota99/ite0/reconsImg.fit /home/yoyisaurio/Desktop/ds9/reconsImg.fit

// sudo scp rarmijo@158.170.35.139:/home/rarmijo/Desktop/proyecto/float_calCompresion_baseNormal_cota99/ite0/reconsImg.fit /home/yoyisaurio/Desktop/ds9/nuevito.fit

// nvcc nuevo.cu -lcudart -lcublas -lcuda -lblasx -I/opt/arrayfire/include/ -L/opt/arrayfire/lib64/ -lafcuda -lcfitsio -o nuevo

struct parametros_BaseRect
{
  float* u;
  float* v;
  float* w;
  float delta_u;
  float delta_v;
  float* matrizDeUnos;
  long cantVisi;
  long N;
  float estrechezDeBorde;
};

struct parametros_BaseNormal
{
  float* u;
  float* v;
  float* w;
  float delta_u;
  float delta_v;
  long cantVisi;
  long N;
};

static int Stopping_Rule(float x0, float x1, float tolerance);

#define sqrt5 2.236067977499789696

char* numAString(int* numero)
{
  int cantCarac = (*numero)/10 + 1;
  char* numComoString = (char*) malloc(sizeof(char)*cantCarac);
  return numComoString;
}

float calPendiente(float* x, int largoDeX, float* y)
{
  float sumadeYs = 0.0;
  float sumadeXs = 0.0;
  float sumaDeLosCuadradosdeXs = 0.0;
  float sumaDeMultdeXsconYs = 0.0;
  for(int i=0; i<largoDeX; i++)
  {
    float xActual = x[i];
    float yActual = y[i];
    sumadeYs += yActual;
    sumadeXs += xActual;
    sumaDeMultdeXsconYs += xActual * yActual;
    sumaDeLosCuadradosdeXs += xActual * xActual;
  }
  float cuadradoDeLaSumadeXs = sumadeXs * sumadeXs;
  float numerador = largoDeX * sumaDeMultdeXsconYs - sumadeXs * sumadeYs;
  float denominador = largoDeX * sumaDeLosCuadradosdeXs - cuadradoDeLaSumadeXs;
  return numerador/denominador;
}

float* linspace(float a, float b, long n)
{
    float c;
    int i;
    float* u;
    hipMallocManaged(&u, n*sizeof(float));
    c = (b - a)/(n - 1);
    for(i = 0; i < n - 1; ++i)
        u[i] = a + i*c;
    u[n - 1] = b;
    return u;
}

void imprimirVector(float* lista, int tamanoLista)
{
  int i;
  for(i=0;i<tamanoLista;i++)
  {
    printf("%f\n",lista[i]);
  }
  printf("\n");
}

void imprimirMatrizColumna(float* vector, long cantFilas, long cantColumnas)
{
  long i,j;
  for(i=0;i<cantFilas;i++)
  {
    for(j=0;j<cantColumnas;j++)
    {
      printf("%.12e ", vector[(((j)*(cantFilas))+(i))]);
    }
    printf("\n");
  }
  printf("\n");
}

void imprimirMatrizPura(float* matriz, int cantFilas, int cantColumnas)
{
  for(int i=0; i<cantFilas; i++)
  {
    for(int j=0; j<cantColumnas; j++)
    {
      printf("%f ", matriz[i*cantColumnas+j]);
    }
    printf("\n");
  }
  printf("\n");
}

void escribirCoefs(float* coefs, char* nombreArchivo, long cantFilas, long cantColumnas)
{
  FILE* archivo = fopen(nombreArchivo, "w");
  for(long i=0;i<cantFilas;i++)
  {
    for(long j=0;j<cantColumnas;j++)
    {
      fprintf(archivo, "%.12e ", coefs[(((j)*(cantFilas))+(i))]);
    }
    fprintf(archivo, "\n");
  }
  fclose(archivo);
}

float** crearMatrizDouble(int cantFilas, int cantColumnas)
{
  float** matriz = (float**) calloc(cantFilas, sizeof(float*));
  int i;
  for(i=0;i<cantFilas;i++)
  {
    matriz[i] = (float*) calloc(cantColumnas, sizeof(float));
  }
  return matriz;
}

void inicializarMatriz(float** matriz, int cantFilas, int cantColumnas)
{
  int i;
  int j;
  int contador = 0;
  for(i=0;i<cantFilas;i++)
  {
      for(j=0;j<cantColumnas;j++)
      {
        matriz[i][j] = contador;
        contador++;
      }
  }
}

float* transformarMatrizAMatrizColumna(float** matriz, int cantFilas, int cantColumnas)
{
  float* nuevoVector = (float*) calloc(cantFilas*cantColumnas,sizeof(float));
  int i,j;
  for(j=0;j<cantColumnas;j++)
  {
    for(i=0;i<cantFilas;i ++)
    {
      nuevoVector[(((j)*(cantFilas))+(i))]= matriz[i][j];
    }
  }
  return nuevoVector;
}

float** transformarMatrizColumnaAMatriz(float* matrizColumna, int cantFilas, int cantColumnas)
{
  float** matriz = crearMatrizDouble(cantFilas,cantColumnas);
  int i,j;
  for(j=0;j<cantColumnas;j++)
  {
    for(i=0;i<cantFilas;i ++)
    {
      matriz[i][j] = matrizColumna[(((j)*(cantFilas))+(i))];
    }
  }
  return matriz;
}

void multMatrices(float* a, long m, long k, float* b, long n, float* c)
{
  hipError_t cudaStat;
  hipblasStatus_t stat;
  cublasXtHandle_t handle;
  stat = cublasXtCreate(&handle);
  int devices[1] = { 0 };
  if(cublasXtDeviceSelect(handle, 1, devices) != HIPBLAS_STATUS_SUCCESS)
  {
    printf("set devices fail\n");
  }
  float al = 1.0;
  float bet = 0.0;
  stat = cublasXtSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,m,n,k,&al,a,m,b,k,&bet,c,m);
  hipDeviceSynchronize();
  for(long i=0; i<m*n;i++)
  {
    if(isnan(c[i]))
    {
      printf("Valor nan encontrado en multMatrices.\n");
      break;
    }
  }
  cublasXtDestroy(handle);
}

// void multMatrices(float* a, long m, long k, float* b, long n, float* c)
// {
//   hipError_t cudaStat;
//   hipblasStatus_t stat;
//   hipblasHandle_t handle;
//   stat = hipblasCreate(&handle);
//   float al = 1.0;
//   float bet = 0.0;
//   stat = hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,m,n,k,&al,a,m,b,k,&bet,c,m);
//   hipDeviceSynchronize();
//   for(long i=0; i<m*n;i++)
//   {
//     if(isnan(c[i]))
//     {
//       printf("Valor nan encontrado en multMatrices.\n");
//       break;
//     }
//   }
//   hipblasDestroy(handle);
// }

// void combinacionLinealMatrices(float al, float* a, long m, long k, float bet, float* c)
// {
//   long n = k;
//   hipError_t cudaStat;
//   hipblasStatus_t stat;
//   cublasXtHandle_t handle;
//   float* b;
//   hipMallocManaged(&b, k*n*sizeof(float));
//   hipMemset(b, 0, k*n*sizeof(float));
//   for(int i=0; i<n; i++)
//   {
//     b[(i*n+i)] = 1.0;
//   }
//   stat = cublasXtCreate(&handle);
//   int devices[1] = { 0 };
//   if(cublasXtDeviceSelect(handle, 1, devices) != HIPBLAS_STATUS_SUCCESS)
//   {
//     printf("set devices fail\n");
//   }
//   stat = cublasXtSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,m,n,k,&al,a,m,b,k,&bet,c,m);
//   hipDeviceSynchronize();
//   for(long i=0; i<m*n;i++)
//   {
//     if(isnan(c[i]))
//     {
//       printf("Valor nan encontrado en combLinealMatrices.\n");
//       break;
//     }
//   }
//   hipFree(b);
//   cublasXtDestroy(handle);
// }

__global__ void multMatrizPorConstante_kernel(float* matrizA, long cantFilas, long cantColumnas, float constante)
{
  long miId = threadIdx.x + blockDim.x * blockIdx.x;
  if(miId < cantFilas*cantColumnas)
  {
    matrizA[miId] = constante * matrizA[miId];
  }
}

void multMatrizPorConstante(float* matrizA, long cantFilasMatrizA, long cantColumnasMatrizA, float constante)
{
  long cantBloques = ceil((float) cantFilasMatrizA*cantColumnasMatrizA/1024);
  multMatrizPorConstante_kernel<<<cantBloques,1024>>>(matrizA, cantFilasMatrizA, cantColumnasMatrizA, constante);
  hipDeviceSynchronize();
}

__global__ void combinacionLinealMatrices_kernel(float al, float* matrizA, long cantFilas, long cantColumnas, float bet, float* matrizB)
{
  long miId = threadIdx.x + blockDim.x * blockIdx.x;
  if(miId < cantFilas*cantColumnas)
  {
    matrizB[miId] = al * matrizA[miId] + bet * matrizB[miId];
  }
}

void combinacionLinealMatrices(float al, float* matrizA, long cantFilas, long cantColumnas, float bet, float* matrizB)
{
  long cantBloques = ceil((float) cantFilas*cantColumnas/1024);
  combinacionLinealMatrices_kernel<<<cantBloques,1024>>>(al, matrizA, cantFilas, cantColumnas, bet, matrizB);
  hipDeviceSynchronize();
}

// void combinacionLinealMatrices(float al, float* a, long m, long k, float bet, float* c)
// {
//   long n = k;
//   hipError_t cudaStat;
//   hipblasStatus_t stat;
//   hipblasHandle_t handle;
//   float* b;
//   hipMallocManaged(&b, k*n*sizeof(float));
//   hipMemset(b, 0, k*n*sizeof(float));
//   for(int i=0; i<n; i++)
//   {
//     b[(i*n+i)] = 1.0;
//   }
//   stat = hipblasCreate(&handle);
//   stat = hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,m,n,k,&al,a,m,b,k,&bet,c,m);
//   hipDeviceSynchronize();
//   for(long i=0; i<m*n;i++)
//   {
//     if(isnan(c[i]))
//     {
//       printf("Valor nan encontrado en combLinealMatrices.\n");
//       break;
//     }
//   }
//   hipFree(b);
//   hipblasDestroy(handle);
// }

// void transponerMatriz(float* matriz, int cantFilas, int cantColumnas, float* matrizTranspuesta)
// {
//   for(int i=0;i<cantFilas;i++)
//   {
//     for(int j=0;j<cantColumnas;j++)
//     {
//       matrizTranspuesta[(((i)*(cantColumnas))+(j))] = matriz[(((j)*(cantFilas))+(i))];
//     }
//   }
// }

// __global__ void transponerMatriz_kernel(float* matrizA, float* matrizA_T, long cantFilas, long cantColumnas)
// {
//   long miId = threadIdx.x + blockDim.x * blockIdx.x * blockDim.x * blockDim.y + blockIdx.y * gridDim.x * blockDim.x * blockDim.y;
//   if(miId < cantFilas*cantColumnas)
//   {
//     long i = miId%cantFilas;
//     long j = miId/cantFilas;
//     matrizA_T[(i*cantColumnas+j)] = matrizA[(j*cantFilas+i)];
//   }
// }

__global__ void transponerMatriz_kernel(float* matrizA, float* matrizA_T, long cantFilas, long cantColumnas)
{
  long miId = threadIdx.x + blockDim.x * blockIdx.x;
  if(miId < cantFilas*cantColumnas)
  {
    long i = miId%cantFilas;
    long j = miId/cantFilas;
    matrizA_T[(i*cantColumnas+j)] = matrizA[(j*cantFilas+i)];
  }
}

void transponerMatriz(float* matrizA, long cantFilasMatrizA, long cantColumnasMatrizA, float* resultado)
{
  long cantBloques = ceil((float) cantFilasMatrizA*cantColumnasMatrizA/1024);
  transponerMatriz_kernel<<<cantBloques,1024>>>(matrizA, resultado, cantFilasMatrizA, cantColumnasMatrizA);
  hipDeviceSynchronize();
}

__global__ void restaVectorColumnaConVector_kernel(float* vectorA, long largoVectorA, float* vectorB, long largoVectorB, float* resultado)
{
  long miId = threadIdx.x + blockDim.x * blockIdx.x;
  if(miId < largoVectorA*largoVectorB)
  {
    long i = miId%largoVectorA;
    long j = miId/largoVectorA;
    resultado[miId] = vectorA[i] - vectorB[j];
  }
}

float* restaVectorColumnaConVector(float* vectorA, long largoVectorA, float* vectorB, long largoVectorB)
{
  float* resultado;
  hipMallocManaged(&resultado,largoVectorA*largoVectorB*sizeof(float));
  long cantBloques = ceil((float) largoVectorA*largoVectorB/1024);
  restaVectorColumnaConVector_kernel<<<cantBloques,1024>>>(vectorA, largoVectorA, vectorB, largoVectorB, resultado);
  hipDeviceSynchronize();
  return resultado;
}

void vectorColumnaAMatriz(float* vectorA, long cantFilas, long cantColumnas, float* nuevaMatriz)
{
  float* vectorDeUnos;
  hipMallocManaged(&vectorDeUnos,cantColumnas*sizeof(float));
  for(long i=0; i<cantColumnas; i++)
  {
    vectorDeUnos[i] = 1.0;
  }
  multMatrices(vectorA, cantFilas, 1, vectorDeUnos, cantColumnas, nuevaMatriz);
  hipFree(vectorDeUnos);
}

__global__ void hadamardProduct_kernel(float* matrizA, float* matrizB, float* resultado, long cantFilas, long cantColumnas)
{
  long miId = threadIdx.x + blockDim.x * blockIdx.x;
  if(miId < cantFilas*cantColumnas)
  {
    resultado[miId] = matrizA[miId]*matrizB[miId];
  }
}

void hadamardProduct(float* matrizA, long cantFilasMatrizA, long cantColumnasMatrizA, float* matrizB, float* resultado)
{
  long cantBloques = ceil((float) cantFilasMatrizA*cantColumnasMatrizA/1024);
  hadamardProduct_kernel<<<cantBloques,1024>>>(matrizA, matrizB, resultado, cantFilasMatrizA, cantColumnasMatrizA);
  hipDeviceSynchronize();
}

float dotProduct(float* x, long n, float* y)
{
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  stat = hipblasCreate(&handle);
  float result;
  stat = hipblasSdot(handle,n,x,1,y,1,&result);
  hipblasDestroy(handle);
  return result;
}

__global__ void calcularExp_kernel(float* a, float* c, long cantFilas, long cantColumnas)
{
  long miId = threadIdx.x + blockDim.x * blockIdx.x;
  if(miId < cantFilas*cantColumnas)
  {
    c[miId] = exp(a[miId]);
  }
}

void calcularExp(float* matrizA, long cantFilasMatrizA, long cantColumnasMatrizA)
{
  long cantBloques = ceil((float) cantFilasMatrizA*cantColumnasMatrizA/1024);
  calcularExp_kernel<<<cantBloques,1024>>>(matrizA, matrizA, cantFilasMatrizA, cantColumnasMatrizA);
  hipDeviceSynchronize();
}

__global__ void calcularInvFrac_kernel(float* a, float* c, long cantFilas, long cantColumnas)
{
  long miId = threadIdx.x + blockDim.x * blockIdx.x;
  if(miId < cantFilas*cantColumnas)
  {
    c[miId] = 1.0/a[miId];
  }
}

void calcularInvFrac(float* matrizA, long cantFilasMatrizA, long cantColumnasMatrizA)
{
  long cantBloques = ceil((float) cantFilasMatrizA*cantColumnasMatrizA/1024);
  calcularInvFrac_kernel<<<cantBloques,1024>>>(matrizA, matrizA, cantFilasMatrizA, cantColumnasMatrizA);
  hipDeviceSynchronize();
}

void calVisModelo(float* MV, long cantFilasMV, long cantColumnasMV, float* MC, long cantColumnasMU, float* MU, float* matrizDeUnosTamN, float* visModelo_paso3)
{
  float* MU_T;
  hipMallocManaged(&MU_T, cantFilasMV*cantColumnasMU*sizeof(float));
  transponerMatriz(MU, cantFilasMV, cantColumnasMU, MU_T);
  float* visModelo_paso1;
  hipMallocManaged(&visModelo_paso1, cantColumnasMV*cantFilasMV*sizeof(float));
  hipMemset(visModelo_paso1, 0, cantColumnasMV*cantFilasMV*sizeof(float));
  multMatrices(MC, cantColumnasMV, cantColumnasMU, MU_T, cantFilasMV, visModelo_paso1);
  hipFree(MU_T);
  float* transpuesta;
  hipMallocManaged(&transpuesta, cantColumnasMV*cantFilasMV*sizeof(float));
  transponerMatriz(visModelo_paso1, cantColumnasMV, cantFilasMV, transpuesta);
  hipFree(visModelo_paso1);
  float* visModelo_paso2;
  hipMallocManaged(&visModelo_paso2, cantFilasMV*cantColumnasMV*sizeof(float));
  hadamardProduct(MV, cantFilasMV, cantColumnasMV, transpuesta, visModelo_paso2);
  hipFree(transpuesta);
  multMatrices(visModelo_paso2, cantFilasMV, cantColumnasMV, matrizDeUnosTamN, 1, visModelo_paso3);
  hipFree(visModelo_paso2);
}

float* calResidual(float* visObs, float* MV, long cantFilasMV, long cantColumnasMV, float* MC, long cantColumnasMU, float* MU, float* matrizDeUnosTamN)
{
  float* visModelo;
  hipMallocManaged(&visModelo, cantFilasMV*sizeof(float));
  hipMemset(visModelo, 0, cantFilasMV*sizeof(float));
  calVisModelo(MV, cantFilasMV, cantColumnasMV, MC, cantColumnasMU, MU, matrizDeUnosTamN, visModelo);
  combinacionLinealMatrices(-1.0, visObs, cantFilasMV, 1, 1.0, visModelo);
  return visModelo;
}

float calCosto(float* residual, long cantVisi, float* w)
{
  float* resultado;
  hipMallocManaged(&resultado, cantVisi*sizeof(float));
  hadamardProduct(residual, cantVisi, 1, w, resultado);
  float total = dotProduct(resultado, cantVisi, residual);
  hipFree(resultado);
  return total;
}

__global__ void MultPorDifer_kernel(float* matrizA, float* matrizB, float* resultado, long cantFilas, long cantColumnas)
{
  long miId = threadIdx.x + blockDim.x * blockIdx.x;
  if(miId < cantFilas*cantColumnas)
  {
    long posicionEnB = miId%cantFilas;
    resultado[miId] = matrizA[miId]*matrizB[posicionEnB];
  }
}

void MultPorDifer(float* matrizA, long cantFilas, long cantColumnas, float* diferencias, float* resultado)
{
  long cantBloques = ceil((float) cantFilas*cantColumnas/1024);
  MultPorDifer_kernel<<<cantBloques,1024>>>(matrizA, diferencias, resultado, cantFilas, cantColumnas);
  hipDeviceSynchronize();
}

void calGradiente(float* residual, float* MV, long cantFilasMV, long cantColumnasMV, float* MU, long cantColumnasMU, float* w, float* total_paso2)
{
  float* diferencia;
  hipMallocManaged(&diferencia, cantFilasMV*sizeof(float));
  hadamardProduct(residual, cantFilasMV, 1, w, diferencia);
  float* total_paso1;
  hipMallocManaged(&total_paso1, cantColumnasMV*cantFilasMV*sizeof(float));
  MultPorDifer(MV, cantFilasMV, cantColumnasMV, diferencia, total_paso1);
  hipFree(diferencia);
  float* total_paso1_5;
  hipMallocManaged(&total_paso1_5, cantColumnasMV*cantFilasMV*sizeof(float));
  transponerMatriz(total_paso1, cantFilasMV, cantColumnasMV, total_paso1_5);
  hipFree(total_paso1);
  multMatrices(total_paso1_5, cantColumnasMV, cantFilasMV, MU, cantColumnasMU, total_paso2);
  hipFree(total_paso1_5);
}

float calAlpha(float* gradiente, long cantFilasMC, long cantColumnasMC, float* pActual, float* MV, long cantFilasMV, long cantColumnasMV, float* MU, long cantColumnasMU, float* w, float* matrizDeUnosTamN, int* flag_NOESPOSIBLEMINIMIZAR)
{
  float* gradienteNegativo;
  hipMallocManaged(&gradienteNegativo, cantFilasMC*cantColumnasMC*sizeof(float));
  hipMemset(gradienteNegativo, 0, cantFilasMC*cantColumnasMC*sizeof(float));
  combinacionLinealMatrices(-1.0, gradiente, cantFilasMC, cantColumnasMC, 0.0, gradienteNegativo);
  float numerador = dotProduct(gradienteNegativo, cantFilasMC*cantColumnasMC, pActual);
  hipFree(gradienteNegativo);
  float* visModeloP;
  hipMallocManaged(&visModeloP, cantFilasMV*sizeof(float));
  hipMemset(visModeloP, 0, cantFilasMV*sizeof(float));
  calVisModelo(MV, cantFilasMV, cantColumnasMV, pActual, cantColumnasMU, MU, matrizDeUnosTamN, visModeloP);
  float* gradP;
  hipMallocManaged(&gradP, cantFilasMC * cantColumnasMC*sizeof(float));
  hipMemset(gradP, 0, cantFilasMC * cantColumnasMC*sizeof(float));
  calGradiente(visModeloP, MV, cantFilasMV, cantColumnasMV, MU, cantColumnasMU, w, gradP);
  hipFree(visModeloP);
  float denominador = dotProduct(pActual, cantFilasMC * cantColumnasMC, gradP);
  hipFree(gradP);
  if(denominador == 0.0)
  {
    *flag_NOESPOSIBLEMINIMIZAR = 1;
  }
  return numerador/denominador;
}

float calBeta_Fletcher_Reeves(float* gradienteActual, long tamanoGradiente, float* gradienteAnterior)
{
  float numerador = dotProduct(gradienteActual, tamanoGradiente, gradienteActual);
  float denominador = dotProduct(gradienteAnterior, tamanoGradiente, gradienteAnterior);
  float resultado = numerador/denominador;
  return resultado;
}

float* calInfoFisherDiag(float* MV, long cantFilasMV, long cantColumnasMV, float* MU, float* w)
{
  float* MV_T;
  hipMallocManaged(&MV_T, cantFilasMV*cantColumnasMV*sizeof(float));
  transponerMatriz(MV, cantFilasMV, cantColumnasMV, MV_T);
  float* primeraMatriz_fase1;
  hipMallocManaged(&primeraMatriz_fase1, cantColumnasMV*cantFilasMV*sizeof(float));
  hadamardProduct(MV_T, cantColumnasMV, cantFilasMV, MV_T, primeraMatriz_fase1);
  hipFree(MV_T);
  float* wMatriz;
  hipMallocManaged(&wMatriz, cantFilasMV*cantColumnasMV*sizeof(float));
  hipMemset(wMatriz, 0, cantFilasMV*cantColumnasMV*sizeof(float));
  vectorColumnaAMatriz(w, cantFilasMV, cantColumnasMV, wMatriz);
  float* wmatriz_T;
  hipMallocManaged(&wmatriz_T, cantFilasMV*cantColumnasMV*sizeof(float));
  transponerMatriz(wMatriz, cantFilasMV, cantColumnasMV, wmatriz_T);
  hipFree(wMatriz);
  float* primeraMatriz_fase2;
  hipMallocManaged(&primeraMatriz_fase2, cantColumnasMV*cantFilasMV*sizeof(float));
  hadamardProduct(primeraMatriz_fase1, cantColumnasMV, cantFilasMV, wmatriz_T, primeraMatriz_fase2);
  hipFree(primeraMatriz_fase1);
  hipFree(wmatriz_T);
  float* MU_T;
  hipMallocManaged(&MU_T, cantFilasMV*cantColumnasMV*sizeof(float));
  transponerMatriz(MU, cantFilasMV, cantColumnasMV, MU_T);
  float* segundaMatriz;
  hipMallocManaged(&segundaMatriz, cantFilasMV*cantColumnasMV*sizeof(float));
  hadamardProduct(MU_T, cantFilasMV, cantColumnasMV, MU_T, segundaMatriz);
  hipFree(MU_T);
  float* resultado_fase1;
  hipMallocManaged(&resultado_fase1, cantColumnasMV*cantFilasMV*sizeof(float));
  hadamardProduct(primeraMatriz_fase2, cantColumnasMV, cantFilasMV, segundaMatriz, resultado_fase1);
  hipFree(primeraMatriz_fase2);
  hipFree(segundaMatriz);
  float* vectorDeUnos;
  hipMallocManaged(&vectorDeUnos, cantFilasMV*sizeof(float));
  float* resultado_fase2;
  hipMallocManaged(&resultado_fase2, cantColumnasMV*sizeof(float));
  hipMemset(resultado_fase2, 0, cantColumnasMV*sizeof(float));
  for(long i=0; i<cantFilasMV; i++)
  {
    vectorDeUnos[i] = 1;
  }
  multMatrices(resultado_fase1, cantColumnasMV, cantFilasMV, vectorDeUnos, 1, resultado_fase2);
  hipFree(resultado_fase1);
  float medidaInfoMaximoDiagonal = 0.0;
  for (long i=0; i<cantColumnasMV; i++)
  {
      if(resultado_fase2[i] > medidaInfoMaximoDiagonal)
        medidaInfoMaximoDiagonal = resultado_fase2[i];
  }
  float medidaInfoSumaDiagonal = dotProduct(resultado_fase2, cantColumnasMV, vectorDeUnos);
  hipFree(vectorDeUnos);
  hipFree(resultado_fase2);
  float* medidasDeInfo = (float*) malloc(sizeof(float)*2);
  medidasDeInfo[0] = medidaInfoSumaDiagonal;
  medidasDeInfo[1] = medidaInfoMaximoDiagonal;
  return medidasDeInfo;
}

float* estimacionDePlanoDeFourier(float* MV, long cantFilasMV, long cantColumnasMV, float* MC, long cantFilasMC, long cantColumnasMC, float* MU)
{
  float* MU_T;
  hipMallocManaged(&MU_T, cantFilasMV*cantColumnasMV*sizeof(float));
  transponerMatriz(MU, cantFilasMV, cantColumnasMV, MU_T);
  float* resultado_paso1;
  hipMallocManaged(&resultado_paso1, cantFilasMC*cantFilasMV*sizeof(float));
  hipMemset(resultado_paso1, 0, cantFilasMC*cantFilasMV*sizeof(float));
  multMatrices(MC, cantFilasMC, cantColumnasMC, MU_T, cantFilasMV, resultado_paso1);
  hipFree(MU_T);
  float* resultado_paso2;
  hipMallocManaged(&resultado_paso2, cantFilasMV*cantFilasMV*sizeof(float));
  hipMemset(resultado_paso2, 0, cantFilasMV*cantFilasMV*sizeof(float));
  multMatrices(MV, cantFilasMV, cantColumnasMV, resultado_paso1, cantFilasMV, resultado_paso2);
  hipFree(resultado_paso1);
  return resultado_paso2;
}

void printerror_cfitsio( int status)
{
    if (status)
    {
       fits_report_error(stderr, status);
       exit( status );
    }
    return;
}

void escribirTransformadaInversaFourier2D(float* estimacionFourier_ParteImag, float* estimacionFourier_ParteReal, long N, char* nombreArchivo)
{
  af::array estimacionFourier_ParteImag_GPU(N, N, estimacionFourier_ParteImag);
  af::array estimacionFourier_ParteReal_GPU(N, N, estimacionFourier_ParteReal);
  af::array mapaFourierRecons = af::complex(estimacionFourier_ParteReal_GPU, estimacionFourier_ParteImag_GPU);
  estimacionFourier_ParteImag_GPU.unlock();
  estimacionFourier_ParteReal_GPU.unlock();
  mapaFourierRecons = af::shift(mapaFourierRecons, (mapaFourierRecons.dims(0)+1)/2, (mapaFourierRecons.dims(1)+1)/2);
  mapaFourierRecons = af::ifft2(mapaFourierRecons, N, N);
  mapaFourierRecons = af::shift(mapaFourierRecons, (mapaFourierRecons.dims(0)+1)/2, (mapaFourierRecons.dims(1)+1)/2);
  mapaFourierRecons = af::real(mapaFourierRecons);
  mapaFourierRecons = af::flip(mapaFourierRecons, 0);
  mapaFourierRecons = af::transpose(mapaFourierRecons);
  float* auxiliar_mapaFourierRecons = mapaFourierRecons.device<float>();
  float* inver_visi = (float*) calloc(N*N, sizeof(float));
  hipMemcpy(inver_visi, auxiliar_mapaFourierRecons, N*N*sizeof(float), hipMemcpyDeviceToHost);
  mapaFourierRecons.unlock();
  fitsfile *fptr;
  int status;
  long fpixel, nelements;
  int bitpix = FLOAT_IMG;
  long naxis = 2;
  long naxes[2] = {N, N};
  remove(nombreArchivo);
  status = 0;
  if (fits_create_file(&fptr, nombreArchivo, &status))
    printerror_cfitsio(status);
  if (fits_create_img(fptr, bitpix, naxis, naxes, &status))
    printerror_cfitsio(status);
  fpixel = 1;
  nelements = naxes[0] * naxes[1];
  if (fits_write_img(fptr, TFLOAT, fpixel, nelements, inver_visi, &status))
    printerror_cfitsio(status);
  if (fits_close_file(fptr, &status))
    printerror_cfitsio(status);
  free(inver_visi);
}

float* calcularMV_Rect(float* v, float delta_v, long cantVisi, long N, float estrechezDeBorde, float ancho, float* matrizDeUnos)
{
  float* desplazamientoEnV = linspace((-N/2.0) * delta_v, ((N/2.0) - 1.0) * delta_v, N);
  float* primeraFraccionV;
  hipMallocManaged(&primeraFraccionV, cantVisi * N * sizeof(float));
  hipMemset(primeraFraccionV, 0, cantVisi * N * sizeof(float));
  float* segundaFraccionV;
  hipMallocManaged(&segundaFraccionV, cantVisi * N * sizeof(float));
  for(long i=0; i<(cantVisi*N); i++)
  {
    segundaFraccionV[i] = 1.0;
  }
  float* matrizDiferenciaV = restaVectorColumnaConVector(v, cantVisi, desplazamientoEnV, N);
  hipFree(desplazamientoEnV);
  combinacionLinealMatrices(-1.0 * estrechezDeBorde, matrizDiferenciaV, cantVisi, N, 0.0, primeraFraccionV);
  combinacionLinealMatrices(estrechezDeBorde, matrizDiferenciaV, cantVisi, N, -1 * estrechezDeBorde * ancho, segundaFraccionV);
  hipFree(matrizDiferenciaV);
  calcularExp(primeraFraccionV, cantVisi, N);
  calcularExp(segundaFraccionV, cantVisi, N);
  combinacionLinealMatrices(1.0, matrizDeUnos, cantVisi, N, 1.0, primeraFraccionV);
  combinacionLinealMatrices(1.0, matrizDeUnos, cantVisi, N, 1.0, segundaFraccionV);
  calcularInvFrac(primeraFraccionV, cantVisi, N);
  calcularInvFrac(segundaFraccionV, cantVisi, N);
  float* MV;
  hipMallocManaged(&MV, cantVisi * N * sizeof(float));
  for(long i=0; i<(cantVisi*N); i++)
  {
    MV[i] = 1.0/ancho;
  }
  combinacionLinealMatrices(1.0, primeraFraccionV, cantVisi, N, 1.0, segundaFraccionV);
  hipFree(primeraFraccionV);
  combinacionLinealMatrices(1.0/ancho, segundaFraccionV, cantVisi, N, -1.0, MV);
  hipFree(segundaFraccionV);
  return MV;
}

float* calcularMV_Rect_estFourier(float ancho, long N, float delta_v, float* matrizDeUnos, float estrechezDeBorde, float* matrizDeUnosEstFourier)
{
  float* coordenadasVCentrosCeldas = linspace((-N/2.0) * delta_v, ((N/2.0) - 1.0) * delta_v, N);
  combinacionLinealMatrices(0.5 * delta_v, matrizDeUnosEstFourier, N, 1, 1.0, coordenadasVCentrosCeldas);
  float* MV_AF = calcularMV_Rect(coordenadasVCentrosCeldas, delta_v, N, N, estrechezDeBorde, ancho, matrizDeUnos);
  hipFree(coordenadasVCentrosCeldas);
  return MV_AF;
}

float* calcularMV_Normal(float* v, float delta_v, long cantVisi, long N, float anchoV)
{
  float* CV;
  hipMallocManaged(&CV, N * sizeof(float));
  for(long i=0;i<N;i++)
  {
    CV[i] = 0.5 * delta_v;
  }
  float* CV_sinescalar = linspace((-N/2.0) * delta_v, ((N/2.0) - 1.0) * delta_v, N);
  combinacionLinealMatrices(1.0, CV_sinescalar, N, 1, 1.0, CV);
  hipFree(CV_sinescalar);
  float* MV = restaVectorColumnaConVector(v, cantVisi, CV, N);
  hipFree(CV);
  multMatrizPorConstante(MV, cantVisi, N, 1.0/anchoV);
  hadamardProduct(MV, cantVisi, N, MV, MV);
  multMatrizPorConstante(MV, cantVisi, N, -0.5);
  calcularExp(MV, cantVisi, N);
  multMatrizPorConstante(MV, cantVisi, N, 1.0/sqrt(2.0 * M_PI * anchoV * anchoV));
  return MV;
}

// float* calcularMV_Normal(float* v, float delta_v, int cantVisi, int N, float anchoV)
// {
//   float* CV = (float*) calloc(N, sizeof(float));
//   float* matrizDeCeros = (float*) calloc(cantVisi * N, sizeof(float));
//   for(int i=0;i<N;i++)
//   {
//     CV[i] = 0.5 * delta_v;
//   }
//   float* CV_sinescalar = linspace((-N/2.0) * delta_v, ((N/2.0) - 1) * delta_v, N);
//   combinacionLinealMatrices(1.0, CV_sinescalar, N, 1, 1.0, CV);
//   free(CV_sinescalar);
//   float* MV = restaVectorColumnaConVector(v, cantVisi, CV, N);
//   free(CV);
//   combinacionLinealMatrices(0.0, matrizDeCeros, cantVisi, N, 1.0/anchoV, MV);
//   hadamardProduct(MV, cantVisi, N, MV, MV);
//   combinacionLinealMatrices(0.0, matrizDeCeros, cantVisi, N, -0.5, MV);
//   calcularExp(MV, cantVisi, N);
//   combinacionLinealMatrices(0.0, matrizDeCeros, cantVisi, N, 1.0/sqrt(2.0 * M_PI * anchoV * anchoV), MV);
//   free(matrizDeCeros);
//   return MV;
// }

float* calcularMV_Normal_estFourier(float anchoV, long N, float delta_v, float* matrizDeUnosEstFourier)
{
  float* coordenadasVCentrosCeldas = linspace((-N/2.0) * delta_v, ((N/2.0) - 1.0) * delta_v, N);
  combinacionLinealMatrices(0.5 * delta_v, matrizDeUnosEstFourier, N, 1, 1.0, coordenadasVCentrosCeldas);
  float* MV_AF = calcularMV_Normal(coordenadasVCentrosCeldas, delta_v, N, N, anchoV);
  hipFree(coordenadasVCentrosCeldas);
  return MV_AF;
}

int calCompresionSegunCota(char* nombreArCoef_comp_imag, char* nombreArCoef_comp_real, float* MC_imag, float* MC_imag_comp, float* MC_real, float* MC_real_comp, long cantFilas, long cantColumnas, float cotaEnergia)
{
  long largo = cantFilas * cantColumnas;
  float* MC_img_cuadrado;
  hipMallocManaged(&MC_img_cuadrado, cantFilas*cantColumnas*sizeof(float));
  float* MC_modulo;
  hipMallocManaged(&MC_modulo, cantFilas*cantColumnas*sizeof(float));
  hadamardProduct(MC_imag, cantFilas, cantColumnas, MC_imag, MC_img_cuadrado);
  hadamardProduct(MC_real, cantFilas, cantColumnas, MC_real, MC_modulo);
  combinacionLinealMatrices(1.0, MC_img_cuadrado, cantFilas, cantColumnas, 1.0, MC_modulo);
  hipFree(MC_img_cuadrado);
  af::array MC_modulo_GPU(cantFilas*cantColumnas, MC_modulo);
  af::array MC_modulo_indicesOrde_GPU(cantFilas*cantColumnas);
  af::array MC_modulo_Orde_GPU(cantFilas*cantColumnas);
  af::sort(MC_modulo_Orde_GPU, MC_modulo_indicesOrde_GPU, MC_modulo_GPU, 0, false);
  float total = af::sum<float>(MC_modulo_GPU);
  MC_modulo_Orde_GPU = MC_modulo_Orde_GPU/total;
  af::eval(MC_modulo_Orde_GPU);
  af::sync();
  float* auxiliar_MC_modulo_Orde_GPU = MC_modulo_Orde_GPU.device<float>();
  float* coefsNormalizados = (float*) calloc(largo, sizeof(float));
  hipMemcpy(coefsNormalizados, auxiliar_MC_modulo_Orde_GPU, cantFilas*cantColumnas*sizeof(float), hipMemcpyDeviceToHost);
  MC_modulo_Orde_GPU.unlock();
  long cantCoefsParaCota = 0;
  float sumador = 0.0;
  for(long i=0; i<largo; i++)
  {
     sumador += coefsNormalizados[i];
     cantCoefsParaCota++;
     if(sumador >= cotaEnergia)
     {
       break;
     }
  }
  hipFree(MC_modulo);
  free(coefsNormalizados);

  MC_modulo_GPU = MC_modulo_indicesOrde_GPU(af::seq(0,(cantCoefsParaCota-1)));
  af::array indRepComp = af::constant(0, largo);
  indRepComp(MC_modulo_GPU) = 1;
  MC_modulo_GPU.unlock();
  MC_modulo_indicesOrde_GPU.unlock();

  af::array MC_imag_GPU(cantFilas*cantColumnas, MC_imag);
  af::array MC_real_GPU(cantFilas*cantColumnas, MC_real);
  MC_imag_GPU = MC_imag_GPU * indRepComp;
  MC_real_GPU = MC_real_GPU * indRepComp;
  af::eval(MC_imag_GPU);
  af::eval(MC_real_GPU);
  af::sync();
  indRepComp.unlock();
  float* auxiliar_MC_imag_GPU = MC_imag_GPU.device<float>();
  float* auxiliar_MC_real_GPU = MC_real_GPU.device<float>();
  hipMemcpy(MC_imag_comp, auxiliar_MC_imag_GPU, cantFilas*cantColumnas*sizeof(float), hipMemcpyDeviceToHost);
  MC_imag_GPU.unlock();
  hipMemcpy(MC_real_comp, auxiliar_MC_real_GPU, cantFilas*cantColumnas*sizeof(float), hipMemcpyDeviceToHost);
  MC_real_GPU.unlock();
  escribirCoefs(MC_imag_comp, nombreArCoef_comp_imag, cantFilas, cantColumnas);
  escribirCoefs(MC_real_comp, nombreArCoef_comp_real, cantFilas, cantColumnas);
  return cantCoefsParaCota;
}

float* minGradConjugado_MinCuadra_escritura(char* nombreArchivoMin, char* nombreArchivoCoefs, float* MV, float* MU, float* visibilidades, float* w, long cantVisi, long N, float* matrizDeUnosTamN, int maxIter, float tol)
{
  int flag_NOESPOSIBLEMINIMIZAR = 0;
  float* MC;
  hipMallocManaged(&MC, N*N*sizeof(float));
  hipMemset(MC, 0, N*N*sizeof(float));
  float* residualInit = calResidual(visibilidades, MV, cantVisi, N, MC, N, MU, matrizDeUnosTamN);
  float* gradienteActual;
  hipMallocManaged(&gradienteActual,N*N*sizeof(float));
  hipMemset(gradienteActual, 0, N*N*sizeof(float));
  float* gradienteAnterior;
  hipMallocManaged(&gradienteAnterior,N*N*sizeof(float));
  hipMemset(gradienteAnterior, 0, N*N*sizeof(float));
  float* pActual;
  hipMallocManaged(&pActual,N*N*sizeof(float));
  hipMemset(pActual, 0, N*N*sizeof(float));
  float costoInicial = calCosto(residualInit, cantVisi, w);
  float costoAnterior = costoInicial;
  float costoActual = costoInicial;
  calGradiente(residualInit, MV, cantVisi, N, MU, N, w, gradienteAnterior);
  hipFree(residualInit);
  // for(int i=0; i<N*N; i++)
  // {
  //   if(gradienteAnterior[i] != 0.0)
  //   {
  //     printf("En la linea %d es %f\n", i, gradienteAnterior[i]);
  //   }
  // }
  // exit(-1);
  combinacionLinealMatrices(-1.0, gradienteAnterior, N, N, 0.0, pActual);
  float diferenciaDeCosto = 1.0;
  int i = 0;
  float alpha = 0.0;
  float epsilon = 1e-10;
  float normalizacion = costoAnterior + costoActual + epsilon;
  FILE* archivoMin = fopen(nombreArchivoMin, "w");
  if(archivoMin == NULL)
  {
       printf("Error al crear o abrir el archivo para almacenar la minimizacion.\n");
       exit(0);
  }
  while(maxIter > i && 2.0 * diferenciaDeCosto > tol * normalizacion)
  {
    alpha = calAlpha(gradienteAnterior, N, N, pActual, MV, cantVisi, N, MU, N, w, matrizDeUnosTamN, &flag_NOESPOSIBLEMINIMIZAR);
    if(flag_NOESPOSIBLEMINIMIZAR == 1)
    {
      break;
    }
    combinacionLinealMatrices(alpha, pActual, N, N, 1.0, MC);
    float* residual = calResidual(visibilidades, MV, cantVisi, N, MC, N, MU, matrizDeUnosTamN);
    costoActual = calCosto(residual, cantVisi, w);
    hipMallocManaged(&gradienteActual,N*N*sizeof(float));
    hipMemset(gradienteActual, 0, N*N*sizeof(float));
    calGradiente(residual, MV, cantVisi, N, MU, N, w, gradienteActual);
    hipFree(residual);
    float beta = calBeta_Fletcher_Reeves(gradienteActual, N*N, gradienteAnterior);
    combinacionLinealMatrices(-1.0, gradienteActual, N, N, beta, pActual);
    diferenciaDeCosto = abs(costoAnterior - costoActual);
    normalizacion = costoAnterior + costoActual + epsilon;
    float otro = costoActual - costoAnterior;
    costoAnterior = costoActual;
    float* auxiliar = gradienteAnterior;
    gradienteAnterior = gradienteActual;
    hipFree(auxiliar);
    i++;
    printf( "En la iteracion %d el valor de la funcion de costos es %f con un z de %.12e la diferencia con respecto al anterior costo es %.12e.\n", i, costoActual, alpha, otro);
    fprintf(archivoMin, "En la iteracion %d el valor de la funcion de costos es %f con un z de %.12e la diferencia con respecto al anterior costo es %.12e.\n", i, costoActual, alpha, otro);
  }
  fclose(archivoMin);
  hipFree(gradienteAnterior);
  hipFree(pActual);
  escribirCoefs(MC, nombreArchivoCoefs, N, N);
  return MC;
}

float* minGradConjugado_MinCuadra(float* MV, float* MU, float* visibilidades, float* w, long cantVisi, long N, float* matrizDeUnosTamN, int maxIter, float tol)
{
  int flag_NOESPOSIBLEMINIMIZAR = 0;
  float* MC;
  hipMallocManaged(&MC, N*N*sizeof(float));
  hipMemset(MC, 0, N*N*sizeof(float));
  float* residualInit = calResidual(visibilidades, MV, cantVisi, N, MC, N, MU, matrizDeUnosTamN);
  float* gradienteActual;
  hipMallocManaged(&gradienteActual,N*N*sizeof(float));
  hipMemset(gradienteActual, 0, N*N*sizeof(float));
  float* gradienteAnterior;
  hipMallocManaged(&gradienteAnterior,N*N*sizeof(float));
  hipMemset(gradienteAnterior, 0, N*N*sizeof(float));
  float* pActual;
  hipMallocManaged(&pActual,N*N*sizeof(float));
  hipMemset(pActual, 0, N*N*sizeof(float));
  float costoInicial = calCosto(residualInit, cantVisi, w);
  float costoAnterior = costoInicial;
  float costoActual = costoInicial;
  calGradiente(residualInit, MV, cantVisi, N, MU, N, w, gradienteAnterior);
  hipFree(residualInit);
  combinacionLinealMatrices(-1.0, gradienteAnterior, N, N, 0.0, pActual);
  float diferenciaDeCosto = 1.0;
  int i = 0;
  float alpha = 0.0;
  float epsilon = 1e-10;
  float normalizacion = costoAnterior + costoActual + epsilon;
  while(maxIter > i && 2.0 * diferenciaDeCosto > tol * normalizacion)
  {
    alpha = calAlpha(gradienteAnterior, N, N, pActual, MV, cantVisi, N, MU, N, w, matrizDeUnosTamN, &flag_NOESPOSIBLEMINIMIZAR);
    if(flag_NOESPOSIBLEMINIMIZAR == 1)
    {
      break;
    }
    combinacionLinealMatrices(alpha, pActual, N, N, 1.0, MC);
    float* residual = calResidual(visibilidades, MV, cantVisi, N, MC, N, MU, matrizDeUnosTamN);
    costoActual = calCosto(residual, cantVisi, w);
    hipMallocManaged(&gradienteActual,N*N*sizeof(float));
    hipMemset(gradienteActual, 0, N*N*sizeof(float));
    calGradiente(residual, MV, cantVisi, N, MU, N, w, gradienteActual);
    hipFree(residual);
    float beta = calBeta_Fletcher_Reeves(gradienteActual, N*N, gradienteAnterior);
    combinacionLinealMatrices(-1.0, gradienteActual, N, N, beta, pActual);
    diferenciaDeCosto = abs(costoAnterior - costoActual);
    normalizacion = costoAnterior + costoActual + epsilon;
    float otro = costoActual - costoAnterior;
    costoAnterior = costoActual;
    float* auxiliar = gradienteAnterior;
    gradienteAnterior = gradienteActual;
    hipFree(auxiliar);
    i++;
    printf( "En la iteracion %d el valor de la funcion de costos es %f con un z de %.12e la diferencia con respecto al anterior costo es %.12e.\n", i, costoActual, alpha, otro);
  }
  hipFree(gradienteAnterior);
  hipFree(pActual);
  return MC;
}

float calculateSD(float* data, float mean, long cantElementos)
{
    float SD = 0.0;
    for (long i = 0; i < cantElementos; i++)
        SD += pow(data[i] - mean, 2);
    return sqrt(SD / 10);
}

float calculoDePSNRDeRecorte(float* estimacionFourier_ParteImag, float* estimacionFourier_ParteReal, long N, char* nombreArchivo, clock_t* tiempoTransInver_MejorCompresion)
{
  int columnaDeInicio = 150;
  int columnaDeTermino = 450;
  int filaDeInicio = 100;
  int filaDeTermino = 400;
  *tiempoTransInver_MejorCompresion = clock();
  af::array estimacionFourier_ParteImag_GPU(N, N, estimacionFourier_ParteImag);
  af::array estimacionFourier_ParteReal_GPU(N, N, estimacionFourier_ParteReal);
  af::array mapaFourierRecons = af::complex(estimacionFourier_ParteReal_GPU, estimacionFourier_ParteImag_GPU);
  estimacionFourier_ParteImag_GPU.unlock();
  estimacionFourier_ParteReal_GPU.unlock();
  mapaFourierRecons = af::shift(mapaFourierRecons, (mapaFourierRecons.dims(0)+1)/2, (mapaFourierRecons.dims(1)+1)/2);
  mapaFourierRecons = af::ifft2(mapaFourierRecons, N, N);
  mapaFourierRecons = af::shift(mapaFourierRecons, (mapaFourierRecons.dims(0)+1)/2, (mapaFourierRecons.dims(1)+1)/2);
  mapaFourierRecons = af::real(mapaFourierRecons);
  *tiempoTransInver_MejorCompresion = clock() - *tiempoTransInver_MejorCompresion;
  mapaFourierRecons = af::flip(mapaFourierRecons, 0);
  mapaFourierRecons = af::transpose(mapaFourierRecons);
  float* auxiliar_mapaFourierRecons = mapaFourierRecons.device<float>();
  float* inver_visi = (float*) calloc(N*N, sizeof(float));
  hipMemcpy(inver_visi, auxiliar_mapaFourierRecons, N*N*sizeof(float), hipMemcpyDeviceToHost);
  mapaFourierRecons.unlock();

  int cantFilasARecorrer = columnaDeTermino - columnaDeInicio + 1;
  int cantColumnasARecorrer = filaDeTermino - filaDeInicio + 1;
  int contador = 0;
  int contadorEleExternos = 0;
  float sumaDeValoresExternos = 0.0;
  float maximoValorInterno = 0;
  float* nuevaImagen = (float*) calloc(cantFilasARecorrer*cantColumnasARecorrer, sizeof(float));
  float* elementosExternos = (float*) calloc(N*N, sizeof(float));
  for(int j=0; j<N; j++)
  {
    for(int i=0; i<N; i++)
    {
      if(columnaDeInicio <= i && i <= columnaDeTermino && filaDeInicio <= j && j <= filaDeTermino)
      {
          nuevaImagen[contador] = inver_visi[i+j*N];
          if(maximoValorInterno < inver_visi[i+j*N])
          {
            maximoValorInterno = inver_visi[i+j*N];
          }
          contador++;
      }
      else
      {
        elementosExternos[contadorEleExternos] = inver_visi[i+j*N];
        sumaDeValoresExternos += elementosExternos[contadorEleExternos];
        contadorEleExternos++;
      }
    }
  }
  float mediaExterna = sumaDeValoresExternos/contadorEleExternos;
  float desvEstandar = calculateSD(elementosExternos, mediaExterna, contadorEleExternos);
  free(elementosExternos);
  float PSNR = maximoValorInterno/desvEstandar;
  // printf("El contador es %d\n", contador);
  // printf("La wea total es %d\n", cantFilasARecorrer*cantColumnasARecorrer);
  // printf("La cantidad de elementos externos es %d\n", contadorEleExternos);

  fitsfile *fptr;
  int status;
  long fpixel, nelements;
  int bitpix = FLOAT_IMG;
  long naxis = 2;
  // long naxes[2] = {cantFilasARecorrer, cantColumnasARecorrer};
  long naxes[2] = {N, N};
  remove(nombreArchivo);
  status = 0;
  if (fits_create_file(&fptr, nombreArchivo, &status))
    printerror_cfitsio(status);
  if (fits_create_img(fptr, bitpix, naxis, naxes, &status))
    printerror_cfitsio(status);
  fpixel = 1;
  nelements = naxes[0] * naxes[1];
  // if (fits_write_img(fptr, TFLOAT, fpixel, nelements, nuevaImagen, &status))
  if (fits_write_img(fptr, TFLOAT, fpixel, nelements, inver_visi, &status))
    printerror_cfitsio(status);
  if (fits_close_file(fptr, &status))
    printerror_cfitsio(status);
  free(inver_visi);
  free(nuevaImagen);
  return PSNR;
}

void calCompSegunAncho_Normal_escritura(char nombreDirPrin[], char* nombreDirSec, float ancho, float cotaEnergia, int iterActual, int maxIter, float tol, float* u, float* v, float* w, float* visi_parteImaginaria, float* visi_parteReal, float delta_u, float delta_v, long cantVisi, long N, float* matrizDeUnosTamN, float estrechezDeBorde)
{
  // ############### CONFIG. DE NOMBRES DE ARCHIVOS  ##############
  char nombreArReconsImg[] = "reconsImg.fit";
  char nombreArReconsCompreImg[] = "reconsCompreImg.fit";
  char nombreArMin_imag[] = "minCoefs_imag.txt";
  char nombreArCoef_imag[] = "coefs_imag.txt";
  char nombreArCoef_comp_imag[] = "coefs_comp_imag.txt";
  char nombreArMin_real[] = "minCoefs_real.txt";
  char nombreArCoef_real[] = "coefs_real.txt";
  char nombreArCoef_comp_real[] = "coefs_comp_real.txt";
  char nombreArInfoCompresion[] = "infoCompre.txt";
  char nombreArInfoTiemposEjecu[] = "infoTiemposEjecu.txt";


  // ############### CALCULO DE MU Y MV - CREACION DE DIRECTORIO SEGUNDARIO  ##############
  printf("...Comenzando calculo de MV...\n");
  clock_t tiempoCalculoMV;
  tiempoCalculoMV = clock();
  float* MV = calcularMV_Normal(v, delta_v, cantVisi, N, ancho);
  tiempoCalculoMV = clock() - tiempoCalculoMV;
  float tiempoTotalCalculoMV = ((float)tiempoCalculoMV)/CLOCKS_PER_SEC;
  printf("Calculo de MV completado.\n");

  printf("...Comenzando calculo de MU...\n");
  clock_t tiempoCalculoMU;
  tiempoCalculoMU = clock();
  float* MU = calcularMV_Normal(u, delta_u, cantVisi, N, ancho);
  tiempoCalculoMU = clock() - tiempoCalculoMU;
  float tiempoTotalCalculoMU = ((float)tiempoCalculoMU)/CLOCKS_PER_SEC;
  printf("Calculo de MU completado.\n");

  char* rutaADirecSec = (char*) malloc(strlen(nombreDirPrin)*strlen(nombreDirSec)*sizeof(char)+sizeof(char)*3);
  strcpy(rutaADirecSec, nombreDirPrin);
  strcat(rutaADirecSec, "/");
  strcat(rutaADirecSec, nombreDirSec);
  if(mkdir(rutaADirecSec, 0777) == -1)
  {
      printf("ERROR: No se pudo crear subdirectorio.");
      printf("PROGRAMA ABORTADO.\n");
      exit(0);
  }
  strcat(rutaADirecSec, "/");


  // ############### MINIMIZACION DE COEFS, PARTE IMAGINARIA  ##############
  char* nombreArchivoMin_imag = (char*) malloc(strlen(nombreDirPrin)*strlen(nombreDirSec)*strlen(nombreArMin_imag)*sizeof(char)+sizeof(char)*3);
  strcpy(nombreArchivoMin_imag, rutaADirecSec);
  strcat(nombreArchivoMin_imag, nombreArMin_imag);
  char* nombreArchivoCoefs_imag = (char*) malloc(strlen(nombreDirPrin)*strlen(nombreDirSec)*strlen(nombreArCoef_imag)*sizeof(char)+sizeof(char)*3);
  strcpy(nombreArchivoCoefs_imag, rutaADirecSec);
  strcat(nombreArchivoCoefs_imag, nombreArCoef_imag);
  printf("...Comenzando minimizacion de coeficientes parte imaginaria...\n");
  clock_t tiempoMinPartImag;
  tiempoMinPartImag = clock();
  float* MC_imag = minGradConjugado_MinCuadra_escritura(nombreArchivoMin_imag, nombreArchivoCoefs_imag, MV, MU, visi_parteImaginaria, w, cantVisi, N, matrizDeUnosTamN, maxIter, tol);
  tiempoMinPartImag = clock() - tiempoMinPartImag;
  float tiempoTotalMinPartImag = ((float)tiempoMinPartImag)/CLOCKS_PER_SEC;
  printf("Proceso de minimizacion de coeficientes parte imaginaria terminado.\n");
  free(nombreArchivoMin_imag);
  free(nombreArchivoCoefs_imag);


  // ############### MINIMIZACION DE COEFS, PARTE REAL  ##############
  char* nombreArchivoMin_real = (char*) malloc(strlen(nombreDirPrin)*strlen(nombreDirSec)*strlen(nombreArMin_real)*sizeof(char)+sizeof(char)*3);
  strcpy(nombreArchivoMin_real, rutaADirecSec);
  strcat(nombreArchivoMin_real, nombreArMin_real);
  char* nombreArchivoCoefs_real = (char*) malloc(strlen(nombreDirPrin)*strlen(nombreDirSec)*strlen(nombreArCoef_real)*sizeof(char)+sizeof(char)*3);
  strcpy(nombreArchivoCoefs_real, rutaADirecSec);
  strcat(nombreArchivoCoefs_real, nombreArCoef_real);
  printf("...Comenzando minimizacion de coeficientes parte real...\n");
  clock_t tiempoMinPartReal;
  tiempoMinPartReal = clock();
  float* MC_real = minGradConjugado_MinCuadra_escritura(nombreArchivoMin_real, nombreArchivoCoefs_real, MV, MU, visi_parteReal, w, cantVisi, N, matrizDeUnosTamN, maxIter, tol);
  tiempoMinPartReal = clock() - tiempoMinPartReal;
  float tiempoTotalMinPartReal = ((float)tiempoMinPartReal)/CLOCKS_PER_SEC;
  printf("Proceso de minimizacion de coeficientes parte real terminado.\n");
  free(nombreArchivoMin_real);
  free(nombreArchivoCoefs_real);


  // ############### CALCULO NIVEL DE INFORMACION ##############
  clock_t tiempoInfo;
  tiempoInfo = clock();
  float* medidasDeInfo = calInfoFisherDiag(MV, cantVisi, N, MU, w);
  tiempoInfo = clock() - tiempoInfo;
  float tiempoTotalInfo = ((float)tiempoInfo)/CLOCKS_PER_SEC;
  hipFree(MU);
  hipFree(MV);


  // ############### CALCULO DE GRADO DE COMPRESION ##############
  float* MC_comp_imag;
  hipMallocManaged(&MC_comp_imag,N*N*sizeof(float));
  hipMemset(MC_comp_imag, 0, N*N*sizeof(float));
  float* MC_comp_real;
  hipMallocManaged(&MC_comp_real,N*N*sizeof(float));
  hipMemset(MC_comp_real, 0, N*N*sizeof(float));
  char* nombreArchivoCoef_comp_imag = (char*) malloc(strlen(nombreDirPrin)*strlen(nombreDirSec)*strlen(nombreArCoef_comp_imag)*sizeof(char)*2);
  strcpy(nombreArchivoCoef_comp_imag, rutaADirecSec);
  strcat(nombreArchivoCoef_comp_imag, nombreArCoef_comp_imag);
  char* nombreArchivoCoef_comp_real = (char*) malloc(strlen(nombreDirPrin)*strlen(nombreDirSec)*strlen(nombreArCoef_comp_real)*sizeof(char)*2);
  strcpy(nombreArchivoCoef_comp_real, rutaADirecSec);
  strcat(nombreArchivoCoef_comp_real, nombreArCoef_comp_real);
  printf("...Comenzando calculo de compresion...\n");
  clock_t tiempoCompresion;
  tiempoCompresion = clock();
  int cantCoefs = calCompresionSegunCota(nombreArchivoCoef_comp_imag, nombreArchivoCoef_comp_real, MC_imag, MC_comp_imag, MC_real, MC_comp_real, N, N, cotaEnergia);
  tiempoCompresion = clock() - tiempoCompresion;
  float tiempoTotalCompresion = ((float)tiempoCompresion)/CLOCKS_PER_SEC;
  printf("Proceso de calculo de compresion terminado.\n");
  free(nombreArchivoCoef_comp_imag);
  free(nombreArchivoCoef_comp_real);
  char* nombreArchivoInfoComp = (char*) malloc(strlen(nombreDirPrin)*strlen(nombreArInfoCompresion)*sizeof(char)+sizeof(char)*2);
  strcpy(nombreArchivoInfoComp, nombreDirPrin);
  strcat(nombreArchivoInfoComp, "/");
  strcat(nombreArchivoInfoComp, nombreArInfoCompresion);
  FILE* archivo = fopen(nombreArchivoInfoComp, "a");
  float nivelDeCompresion = 1.0 - 1.0 * cantCoefs / N*N;
  fprintf(archivo, "%d %.12f %12.f %.12e %.12e %.12f %.12d\n", iterActual, ancho/delta_u, ancho, medidasDeInfo[0], medidasDeInfo[1], nivelDeCompresion, cantCoefs);
  fclose(archivo);
  free(nombreArchivoInfoComp);
  free(medidasDeInfo);


  // ############### RECONSTRUCCION DEL PLANO GRILLEADO Y ALMACENAMIENTO DE LA RECONSTRUCCION DE LA IMAGEN ##############
  char* nombreArchivoReconsImg = (char*) malloc(strlen(nombreDirPrin)*strlen(nombreDirSec)*strlen(nombreArReconsImg)*sizeof(char)+sizeof(char)*3);
  strcpy(nombreArchivoReconsImg, rutaADirecSec);
  strcat(nombreArchivoReconsImg, nombreArReconsImg);
  clock_t tiempoCalculoMV_AF;
  tiempoCalculoMV_AF = clock();
  float* MV_AF = calcularMV_Normal_estFourier(ancho, N, delta_v, matrizDeUnosTamN);
  tiempoCalculoMV_AF = clock() - tiempoCalculoMV_AF;
  float tiempoTotalCalculoMV_AF = ((float)tiempoCalculoMV_AF)/CLOCKS_PER_SEC;
  clock_t tiempoCalculoMU_AF;
  tiempoCalculoMU_AF = clock();
  float* MU_AF = calcularMV_Normal_estFourier(ancho, N, delta_u, matrizDeUnosTamN);
  tiempoCalculoMU_AF = clock() - tiempoCalculoMU_AF;
  float tiempoTotalCalculoMU_AF = ((float)tiempoCalculoMU_AF)/CLOCKS_PER_SEC;
  clock_t tiempoReconsFourierPartImag;
  tiempoReconsFourierPartImag = clock();
  float* estimacionFourier_ParteImag = estimacionDePlanoDeFourier(MV_AF, N, N, MC_imag, N, N, MU_AF);
  tiempoReconsFourierPartImag = clock() - tiempoReconsFourierPartImag;
  float tiempoTotalReconsFourierPartImag = ((float)tiempoReconsFourierPartImag)/CLOCKS_PER_SEC;
  hipFree(MC_imag);
  clock_t tiempoReconsFourierPartReal;
  tiempoReconsFourierPartReal = clock();
  float* estimacionFourier_ParteReal = estimacionDePlanoDeFourier(MV_AF, N, N, MC_real, N, N, MU_AF);
  tiempoReconsFourierPartReal = clock() - tiempoReconsFourierPartReal;
  float tiempoTotalReconsFourierPartReal = ((float)tiempoReconsFourierPartReal)/CLOCKS_PER_SEC;
  hipFree(MC_real);
  clock_t tiempoReconsTransInver;
  tiempoReconsTransInver = clock();
  escribirTransformadaInversaFourier2D(estimacionFourier_ParteImag, estimacionFourier_ParteReal, N, nombreArchivoReconsImg);
  tiempoReconsTransInver = clock() - tiempoReconsTransInver;
  float tiempoTotalReconsTransInver = ((float)tiempoReconsTransInver)/CLOCKS_PER_SEC;
  hipFree(estimacionFourier_ParteImag);
  hipFree(estimacionFourier_ParteReal);
  free(nombreArchivoReconsImg);


  // ############### RECONSTRUCCION DEL PLANO GRILLEADO Y ALMACENAMIENTO DE LA RECONSTRUCCION COMPRIMIDA DE LA IMAGEN ##############
  char* nombreArchivoReconsImgComp = (char*) malloc(strlen(nombreDirPrin)*strlen(nombreDirSec)*strlen(nombreArReconsCompreImg)*sizeof(char)+sizeof(char)*3);
  strcpy(nombreArchivoReconsImgComp, rutaADirecSec);
  strcat(nombreArchivoReconsImgComp, nombreArReconsCompreImg);
  clock_t tiempoReconsFourierPartImagComp;
  tiempoReconsFourierPartImagComp = clock();
  float* estimacionFourier_compre_ParteImag = estimacionDePlanoDeFourier(MV_AF, N, N, MC_comp_imag, N, N, MU_AF);
  tiempoReconsFourierPartImagComp = clock() - tiempoReconsFourierPartImagComp;
  float tiempoTotalReconsFourierPartImagComp = ((float)tiempoReconsFourierPartImagComp)/CLOCKS_PER_SEC;
  hipFree(MC_comp_imag);
  clock_t tiempoReconsFourierPartRealComp;
  tiempoReconsFourierPartRealComp = clock();
  float* estimacionFourier_compre_ParteReal = estimacionDePlanoDeFourier(MV_AF, N, N, MC_comp_real, N, N, MU_AF);
  tiempoReconsFourierPartRealComp = clock() - tiempoReconsFourierPartRealComp;
  float tiempoTotalReconsFourierPartRealComp = ((float)tiempoReconsFourierPartRealComp)/CLOCKS_PER_SEC;
  hipFree(MC_comp_real);
  clock_t tiempoReconsTransInverComp;
  tiempoReconsTransInverComp = clock();
  escribirTransformadaInversaFourier2D(estimacionFourier_compre_ParteImag, estimacionFourier_compre_ParteReal, N, nombreArchivoReconsImgComp);
  tiempoReconsTransInverComp = clock() - tiempoReconsTransInverComp;
  float tiempoTotalReconsTransInverComp = ((float)tiempoReconsTransInverComp)/CLOCKS_PER_SEC;
  hipFree(estimacionFourier_compre_ParteImag);
  hipFree(estimacionFourier_compre_ParteReal);
  free(nombreArchivoReconsImgComp);
  hipFree(MU_AF);
  hipFree(MV_AF);


  // ############### ESCRITURA DE ARCHIVO CON TIEMPOS DE EJECUCION ##############
  char* nombreArchivoInfoTiemposEjecu = (char*) malloc(strlen(nombreDirPrin)*strlen(nombreArInfoTiemposEjecu)*sizeof(char)+sizeof(char)*2);
  strcpy(nombreArchivoInfoTiemposEjecu, nombreDirPrin);
  strcat(nombreArchivoInfoTiemposEjecu, "/");
  strcat(nombreArchivoInfoTiemposEjecu, nombreArInfoTiemposEjecu);
  FILE* archivoInfoTiemposEjecu = fopen(nombreArchivoInfoTiemposEjecu, "a");
  fprintf(archivoInfoTiemposEjecu, "%d %.12f %.12e %.12e %.12e %.12e %.12e %.12e %.12e %.12e %.12e %.12e %.12e %.12e %.12e %.12e\n", iterActual, ancho, tiempoTotalCalculoMV, tiempoTotalCalculoMU, tiempoTotalMinPartImag, tiempoTotalMinPartReal, tiempoTotalInfo, tiempoTotalCompresion, tiempoTotalCalculoMV_AF, tiempoTotalCalculoMU_AF, tiempoTotalReconsFourierPartImag, tiempoTotalReconsFourierPartReal, tiempoTotalReconsTransInver, tiempoTotalReconsFourierPartImagComp, tiempoTotalReconsFourierPartRealComp, tiempoTotalReconsTransInverComp);
  fclose(archivoInfoTiemposEjecu);
  free(nombreArchivoInfoComp);

  free(rutaADirecSec);
}

float calPSNRDeDistintasCompresiones(float inicioIntervalo, float finIntervalo, int cantParamEvaInfo, char rutaADirecSec[], char rutaADirecTer[], char nombreArReconsCompreImg[], float* MC_imag, float* MC_real, float* MV_AF, float* MU_AF, long N, clock_t* tiempoReconsParteImag_MejorCompresion, clock_t* tiempoReconsParteReal_MejorCompresion, clock_t* tiempoTransInver_MejorCompresion)
{
  float cotaMinPSNR = 0.75;
  float cotaMinCompresion = 0.2;
  // float limiteInferior = 0.3;
  // float limiteSuperior = 0.8;
  float* datosDelMin = (float*) malloc(sizeof(float)*4);
  long cantCoefsMejorCompre = 0;
  char nombreArchivoTXTCompre[] = "compresiones.txt";
  char nombreArchivoDatosMinPSNR[] = "mejorTradeOffPSNRCompre.txt";
  char nombreArchivoCompreImg[] = "compreImg";


  float* paramEvaInfo = linspace(inicioIntervalo/100.0, finIntervalo/100.0, cantParamEvaInfo);
  float* MC_comp_imag;
  hipMallocManaged(&MC_comp_imag,N*N*sizeof(float));
  hipMemset(MC_comp_imag, 0, N*N*sizeof(float));
  float* MC_comp_real;
  hipMallocManaged(&MC_comp_real,N*N*sizeof(float));
  hipMemset(MC_comp_real, 0, N*N*sizeof(float));
  long largo = N * N;
  float* MC_img_cuadrado;
  hipMallocManaged(&MC_img_cuadrado, N*N*sizeof(float));
  float* MC_modulo;
  hipMallocManaged(&MC_modulo, N*N*sizeof(float));
  hadamardProduct(MC_imag, N, N, MC_imag, MC_img_cuadrado);
  hadamardProduct(MC_real, N, N, MC_real, MC_modulo);
  combinacionLinealMatrices(1.0, MC_img_cuadrado, N, N, 1.0, MC_modulo);
  hipFree(MC_img_cuadrado);
  af::array MC_modulo_GPU(N*N, MC_modulo);
  hipFree(MC_modulo);
  af::array MC_modulo_indicesOrde_GPU(N*N);
  af::array MC_modulo_Orde_GPU(N*N);
  af::sort(MC_modulo_Orde_GPU, MC_modulo_indicesOrde_GPU, MC_modulo_GPU, 0, false);
  float total = af::sum<float>(MC_modulo_GPU);
  MC_modulo_Orde_GPU = MC_modulo_Orde_GPU/total;
  af::eval(MC_modulo_Orde_GPU);
  af::eval(MC_modulo_indicesOrde_GPU);
  af::sync();
  float* auxiliar_MC_modulo_Orde_GPU = MC_modulo_Orde_GPU.device<float>();
  float* auxiliar_MC_modulo_indicesOrde_GPU = MC_modulo_indicesOrde_GPU.device<float>();
  float* coefsNormalizados = (float*) malloc(largo*sizeof(float));
  hipMemcpy(coefsNormalizados, auxiliar_MC_modulo_Orde_GPU, N*N*sizeof(float), hipMemcpyDeviceToHost);
  int* MC_modulo_indicesOrde_CPU = (int*) malloc(largo*sizeof(int));
  hipMemcpy(MC_modulo_indicesOrde_CPU, auxiliar_MC_modulo_indicesOrde_GPU, N*N*sizeof(int), hipMemcpyDeviceToHost);
  MC_modulo_Orde_GPU.unlock();
  MC_modulo_GPU.unlock();
  MC_modulo_indicesOrde_GPU.unlock();
  long cantCoefsParaCota = 0;
  float sumador = 0.0;
  long iExterno = 0;
  float* cantidadPorcentualDeCoefs = linspace(0.0, largo, largo+1);
  combinacionLinealMatrices(0.0, cantidadPorcentualDeCoefs, largo+1, 1, 1.0/largo, cantidadPorcentualDeCoefs);
  char* nombreArchivoCompresiones = (char*) malloc(sizeof(char)*strlen(rutaADirecSec)*strlen(nombreArchivoTXTCompre)+sizeof(char)*4);
  strcpy(nombreArchivoCompresiones, rutaADirecSec);
  strcat(nombreArchivoCompresiones, "/");
  strcat(nombreArchivoCompresiones, nombreArchivoTXTCompre);
  FILE* archivoPSNR = fopen(nombreArchivoCompresiones, "a");
  float* vectorDePSNR = (float*) calloc(cantParamEvaInfo, sizeof(float));
  float* porcenReal = (float*) calloc(cantParamEvaInfo, sizeof(float));
  float* porcenIdeal = (float*) calloc(cantParamEvaInfo, sizeof(float));
  long* cantCoefsUsadas = (long*) calloc(cantParamEvaInfo, sizeof(long));
  float* vectorDePorcenEnergia = (float*) calloc(cantParamEvaInfo, sizeof(float));
  float* vectorDeDifePSNREntrePtosAdya = (float*) calloc(cantParamEvaInfo, sizeof(float));
  int flag_inicioDeVentana = 1;
  int cantPtsVentana = 0;
  int inicioDeVentana = 0;
  clock_t tiempoCualquiera;
  for(long j=0; j<cantParamEvaInfo; j++)
  {
    sumador = 0.0;
    cantCoefsParaCota = 0;
    iExterno = 0;
    for(long i=0; i<largo+1; i++)
    {
      if(cantidadPorcentualDeCoefs[i] < paramEvaInfo[cantParamEvaInfo-1-j])
      {
        sumador += coefsNormalizados[i];
        cantCoefsParaCota++;
      }
      else
      {
        iExterno = i;
        printf("Del %f%% solicitado, el mas cercano correspondiente al %f%% de coefs, lo que corresponde a %ld coeficientes los cuales poseen el %f%% de la energia.\n", paramEvaInfo[cantParamEvaInfo-1-j] * 100, cantidadPorcentualDeCoefs[i] * 100, cantCoefsParaCota, sumador * 100);
        break;
      }
    }
    if(cantCoefsParaCota != 0)
    {
      int* indicesATomar_CPU = (int*) calloc(cantCoefsParaCota, sizeof(int));
      for(int k=0; k<cantCoefsParaCota; k++)
      {
        indicesATomar_CPU[k] = MC_modulo_indicesOrde_CPU[k];
      }
      af::array indicesATomar_GPU(cantCoefsParaCota, indicesATomar_CPU);
      free(indicesATomar_CPU);
      af::array indRepComp = af::constant(0, largo);
      indRepComp(indicesATomar_GPU) = 1;
      indicesATomar_GPU.unlock();
      af::array MC_imag_GPU(N*N, MC_imag);
      af::array MC_real_GPU(N*N, MC_real);
      MC_imag_GPU = MC_imag_GPU * indRepComp;
      MC_real_GPU = MC_real_GPU * indRepComp;
      af::eval(MC_imag_GPU);
      af::eval(MC_real_GPU);
      af::sync();
      indRepComp.unlock();
      float* auxiliar_MC_imag_GPU = MC_imag_GPU.device<float>();
      float* auxiliar_MC_real_GPU = MC_real_GPU.device<float>();
      hipMemcpy(MC_comp_imag, auxiliar_MC_imag_GPU, N*N*sizeof(float), hipMemcpyDeviceToHost);
      MC_imag_GPU.unlock();
      hipMemcpy(MC_comp_real, auxiliar_MC_real_GPU, N*N*sizeof(float), hipMemcpyDeviceToHost);
      MC_real_GPU.unlock();
      float* estimacionFourier_compre_ParteImag = estimacionDePlanoDeFourier(MV_AF, N, N, MC_comp_imag, N, N, MU_AF);
      float* estimacionFourier_compre_ParteReal = estimacionDePlanoDeFourier(MV_AF, N, N, MC_comp_real, N, N, MU_AF);
      int numero = j+1;
      char* numComoString = numAString(&numero);
      sprintf(numComoString, "%d", numero);
      char* nombreArchivoReconsImgComp = (char*) malloc(sizeof(char)*strlen(rutaADirecTer)*strlen(numComoString)*strlen(nombreArchivoCompreImg)+sizeof(char)*7);
      strcpy(nombreArchivoReconsImgComp, rutaADirecTer);
      strcat(nombreArchivoReconsImgComp, "/");
      strcat(nombreArchivoReconsImgComp, nombreArchivoCompreImg);
      strcat(nombreArchivoReconsImgComp, "_");
      strcat(nombreArchivoReconsImgComp, numComoString);
      strcat(nombreArchivoReconsImgComp, ".fit");
      float PSNRActual = calculoDePSNRDeRecorte(estimacionFourier_compre_ParteImag, estimacionFourier_compre_ParteReal, N, nombreArchivoReconsImgComp, &tiempoCualquiera);
      porcenIdeal[j] = 1-paramEvaInfo[cantParamEvaInfo-1-j];
      vectorDePSNR[j] = PSNRActual;
      porcenReal[j] = 1-cantidadPorcentualDeCoefs[iExterno];
      cantCoefsUsadas[j] = cantCoefsParaCota;
      vectorDePorcenEnergia[j] = sumador;
      fprintf(archivoPSNR, "%f %f %f\n", 1-cantidadPorcentualDeCoefs[iExterno], 1-paramEvaInfo[cantParamEvaInfo-1-j], PSNRActual);
      hipFree(estimacionFourier_compre_ParteImag);
      hipFree(estimacionFourier_compre_ParteReal);
      free(numComoString);
      free(nombreArchivoReconsImgComp);
    }
  }
  fclose(archivoPSNR);

  float* vectorDePSNRFiltrado = (float*) calloc(cantParamEvaInfo, sizeof(float));
  gsl_vector* vectorDePSNREnGSL = gsl_vector_alloc(cantParamEvaInfo);
  gsl_vector* vectorDePSNREnGSLFiltrado = gsl_vector_alloc(cantParamEvaInfo);
  for(int i=0; i<cantParamEvaInfo; i++)
  {
    gsl_vector_set(vectorDePSNREnGSL, i, vectorDePSNR[i]);
  }
  gsl_filter_gaussian_workspace* gauss_p = gsl_filter_gaussian_alloc(5);
  gsl_filter_gaussian(GSL_FILTER_END_PADVALUE, 1.0, 0, vectorDePSNREnGSL, vectorDePSNREnGSLFiltrado, gauss_p);
  for(int i=0; i<cantParamEvaInfo; i++)
  {
    vectorDePSNRFiltrado[i] = gsl_vector_get(vectorDePSNREnGSLFiltrado, i);
  }
  gsl_vector_free(vectorDePSNREnGSL);
  gsl_vector_free(vectorDePSNREnGSLFiltrado);
  gsl_filter_gaussian_free(gauss_p);


  // float* listaDeMetricas = (float*) malloc(sizeof(float)*cantParamEvaInfo);
  // float* primeraRecta_subListaDeX = (float*) calloc(cantParamEvaInfo, sizeof(float));
  // float* primeraRecta_subListaDeY = (float*) calloc(cantParamEvaInfo, sizeof(float));
  // float* segundaRecta_subListaDeX = (float*) calloc(cantParamEvaInfo, sizeof(float));
  // float* segundaRecta_subListaDeY = (float*) calloc(cantParamEvaInfo, sizeof(float));
  // memcpy(segundaRecta_subListaDeX, porcenReal, sizeof(float)*cantParamEvaInfo);
  // memcpy(segundaRecta_subListaDeY, vectorDePSNRFiltrado, sizeof(float)*cantParamEvaInfo);
  // primeraRecta_subListaDeX[0] = porcenReal[0];
  // primeraRecta_subListaDeY[0] = vectorDePSNRFiltrado[0];
  // float metricaMin;
  // float metricaActual;
  // int flagPrimerValorParaMetricaMin = 0;
  // printf("7\n");
  // for(int i=1; i<cantParamEvaInfo-1; i++)
  // {
  //     primeraRecta_subListaDeX[i] = porcenReal[i];
  //     primeraRecta_subListaDeY[i] = vectorDePSNRFiltrado[i];
  //     float pendienteDePrimeraRecta = calPendiente(primeraRecta_subListaDeX, i+1, primeraRecta_subListaDeY);
  //     segundaRecta_subListaDeX[i-1] = 0.0;
  //     segundaRecta_subListaDeY[i-1] = 0.0;
  //     float pendienteDeSegundaRecta = calPendiente(&(segundaRecta_subListaDeX[i]), cantParamEvaInfo-i, &(segundaRecta_subListaDeY[i]));
  //     metricaActual = -1.0 * pendienteDeSegundaRecta/pendienteDePrimeraRecta;
  //     listaDeMetricas[i] = metricaActual;
  //     if(limiteInferior <= porcenReal[i] && porcenReal[i] <= limiteSuperior)
  //     {
  //       if(flagPrimerValorParaMetricaMin == 0)
  //       {
  //         metricaMin = metricaActual;
  //         datosDelMin[0] = porcenIdeal[i];
  //         datosDelMin[1] = porcenReal[i];
  //         cantCoefsMejorCompre = cantCoefsUsadas[i];
  //         datosDelMin[2] = vectorDePorcenEnergia[i];
  //         datosDelMin[3] = vectorDePSNR[i];
  //         flagPrimerValorParaMetricaMin = 1;
  //       }
  //       if(metricaActual < metricaMin)
  //       {
  //         metricaMin = metricaActual;
  //         datosDelMin[0] = porcenIdeal[i];
  //         datosDelMin[1] = porcenReal[i];
  //         cantCoefsMejorCompre = cantCoefsUsadas[i];
  //         datosDelMin[2] = vectorDePorcenEnergia[i];
  //         datosDelMin[3] = vectorDePSNR[i];
  //       }
  //     }
  // }

  FILE* archivoRandom = fopen("wea.txt", "w");
  for(int i=0; i<cantParamEvaInfo; i++)
  {
      fprintf(archivoRandom, "%f\n", vectorDePSNRFiltrado[i]);
  }
  fclose(archivoRandom);

  // free(vectorDePSNRFiltrado);
  // free(primeraRecta_subListaDeX);
  // free(primeraRecta_subListaDeY);
  // free(segundaRecta_subListaDeX);
  // free(segundaRecta_subListaDeY);
  // free(porcenIdeal);
  // free(porcenReal);
  // free(cantCoefsUsadas);
  // free(vectorDePorcenEnergia);
  // free(vectorDePSNR);


  for(int j=0; j<cantParamEvaInfo; j++)
  {
    float porcenActual = porcenReal[j];
    float porcenDifActual = vectorDePSNRFiltrado[j]/vectorDePSNRFiltrado[0];
    if(j >= 1)
    {
      if(porcenActual >= cotaMinCompresion && porcenDifActual >= cotaMinPSNR)
      {
        if(flag_inicioDeVentana)
        {
          inicioDeVentana = j;
          flag_inicioDeVentana = 0;
        }
        vectorDeDifePSNREntrePtosAdya[cantPtsVentana] = vectorDePSNRFiltrado[j] - vectorDePSNRFiltrado[j-1];
        // printf("%.12e\n", vectorDeDifePSNREntrePtosAdya[cantPtsVentana]);
        cantPtsVentana++;
      }
    }
  }

  af::array vectorDeDifePSNREntrePtosAdya_GPU(cantPtsVentana, vectorDeDifePSNREntrePtosAdya);
  free(vectorDeDifePSNREntrePtosAdya);
  af::array vectorDeDifePSNREntrePtosAdya_indicesOrde_GPU(cantPtsVentana);
  af::array vectorDeDifePSNREntrePtosAdya_Orde_GPU(cantPtsVentana);
  af::sort(vectorDeDifePSNREntrePtosAdya_Orde_GPU, vectorDeDifePSNREntrePtosAdya_indicesOrde_GPU, vectorDeDifePSNREntrePtosAdya_GPU, 0, true);
  vectorDeDifePSNREntrePtosAdya_GPU.unlock();
  vectorDeDifePSNREntrePtosAdya_Orde_GPU.unlock();
  int* auxiliar_vectorDeDifePSNREntrePtosAdya_indicesOrde_GPU = vectorDeDifePSNREntrePtosAdya_indicesOrde_GPU.device<int>();
  int* vectorDeDifePSNREntrePtosAdya_indicesOrde_CPU = (int*) malloc(sizeof(int)*cantPtsVentana);
  hipMemcpy(vectorDeDifePSNREntrePtosAdya_indicesOrde_CPU, auxiliar_vectorDeDifePSNREntrePtosAdya_indicesOrde_GPU, cantPtsVentana*sizeof(int), hipMemcpyDeviceToHost);
  vectorDeDifePSNREntrePtosAdya_indicesOrde_GPU.unlock();
  // int indiceElegido = vectorDeDifePSNREntrePtosAdya_indicesOrde_CPU[0] + inicioDeVentana - 1;
  printf("El indice elegido es %d\n", indiceElegido);
  free(vectorDeDifePSNREntrePtosAdya_indicesOrde_CPU);
  datosDelMin[0] = porcenIdeal[indiceElegido];
  datosDelMin[1] = porcenReal[indiceElegido];
  cantCoefsMejorCompre = cantCoefsUsadas[indiceElegido];
  datosDelMin[2] = vectorDePorcenEnergia[indiceElegido];
  datosDelMin[3] = vectorDePSNR[indiceElegido];
  free(vectorDePSNRFiltrado);
  free(porcenIdeal);
  free(porcenReal);
  free(cantCoefsUsadas);
  free(vectorDePorcenEnergia);
  free(vectorDePSNR);


  char* nombreArchivoMejorCompre = (char*) malloc(sizeof(char)*strlen(rutaADirecSec)*strlen(nombreArchivoDatosMinPSNR)+sizeof(char)*4);
  strcpy(nombreArchivoMejorCompre, rutaADirecSec);
  strcat(nombreArchivoMejorCompre, "/");
  strcat(nombreArchivoMejorCompre, nombreArchivoDatosMinPSNR);
  FILE* archivoMejorCompre = fopen(nombreArchivoMejorCompre, "w");
  fprintf(archivoMejorCompre, "El tradeoff seleccionado corresponde al %f%% de coefs, el mas cercano correspondiente al %f%% de coefs, lo que corresponde a %ld coeficientes los cuales poseen el %f%% de la energia y un PSNR de %f%%.\n", datosDelMin[0]  * 100, datosDelMin[1]  * 100, cantCoefsMejorCompre, datosDelMin[2]  * 100, datosDelMin[3]);
  free(nombreArchivoMejorCompre);
  free(datosDelMin);
  fclose(archivoMejorCompre);
  float* indicesATomar_CPU = (float*) malloc(cantCoefsMejorCompre*sizeof(float));
  for(int k=0; k<cantCoefsMejorCompre; k++)
  {
    indicesATomar_CPU[k] = MC_modulo_indicesOrde_CPU[k];
  }
  af::array indicesATomar_GPU(cantCoefsMejorCompre, indicesATomar_CPU);
  free(indicesATomar_CPU);
  af::array indRepComp = af::constant(0, largo);
  indRepComp(indicesATomar_GPU) = 1;
  indicesATomar_GPU.unlock();
  af::array MC_imag_GPU(N*N, MC_imag);
  af::array MC_real_GPU(N*N, MC_real);
  MC_imag_GPU = MC_imag_GPU * indRepComp;
  MC_real_GPU = MC_real_GPU * indRepComp;
  af::eval(MC_imag_GPU);
  af::eval(MC_real_GPU);
  af::sync();
  indRepComp.unlock();
  float* auxiliar_MC_imag_GPU = MC_imag_GPU.device<float>();
  float* auxiliar_MC_real_GPU = MC_real_GPU.device<float>();
  hipMemcpy(MC_comp_imag, auxiliar_MC_imag_GPU, N*N*sizeof(float), hipMemcpyDeviceToHost);
  MC_imag_GPU.unlock();
  hipMemcpy(MC_comp_real, auxiliar_MC_real_GPU, N*N*sizeof(float), hipMemcpyDeviceToHost);
  MC_real_GPU.unlock();
  *tiempoReconsParteImag_MejorCompresion = clock();
  float* estimacionFourier_compre_ParteImag = estimacionDePlanoDeFourier(MV_AF, N, N, MC_comp_imag, N, N, MU_AF);
  *tiempoReconsParteImag_MejorCompresion = clock() - *tiempoReconsParteImag_MejorCompresion;
  *tiempoReconsParteReal_MejorCompresion  = clock();
  float* estimacionFourier_compre_ParteReal = estimacionDePlanoDeFourier(MV_AF, N, N, MC_comp_real, N, N, MU_AF);
  *tiempoReconsParteReal_MejorCompresion = clock() - *tiempoReconsParteReal_MejorCompresion;
  char* nombreArchivoReconsImgComp = (char*) malloc(sizeof(char)*strlen(rutaADirecSec)*strlen(nombreArReconsCompreImg)+sizeof(char)*4);
  strcpy(nombreArchivoReconsImgComp, rutaADirecSec);
  strcat(nombreArchivoReconsImgComp, "/");
  strcat(nombreArchivoReconsImgComp, nombreArReconsCompreImg);
  float PSNRActual = calculoDePSNRDeRecorte(estimacionFourier_compre_ParteImag, estimacionFourier_compre_ParteReal, N, nombreArchivoReconsImgComp, tiempoTransInver_MejorCompresion);
  hipFree(estimacionFourier_compre_ParteImag);
  hipFree(estimacionFourier_compre_ParteReal);

  hipFree(MC_comp_imag);
  hipFree(MC_comp_real);
  hipFree(cantidadPorcentualDeCoefs);
  hipFree(paramEvaInfo);
  hipFree(MU_AF);
  hipFree(MV_AF);
  free(coefsNormalizados);
  free(MC_modulo_indicesOrde_CPU);
  free(nombreArchivoCompresiones);
  return cantCoefsMejorCompre;
}

void calCompSegunAncho_Rect_escritura(char nombreDirPrin[], char* nombreDirSec, char nombreDirTer[], float ancho, float cotaEnergia, int iterActual, int maxIter, float tol, float* u, float* v, float* w, float* visi_parteImaginaria, float* visi_parteReal, float delta_u, float delta_v, float* matrizDeUnos, long cantVisi, long N, float* matrizDeUnosTamN, float estrechezDeBorde)
{
  float inicioPorcenCompre = 0.0;
  float terminoPorcenCompre = 0.2;
  int cantPorcen = 101;
  // int cantPorcen = 2;


  // ############### CONFIG. DE NOMBRES DE ARCHIVOS  ##############
  char nombreArReconsImg[] = "reconsImg.fit";
  char nombreArReconsCompreImg[] = "reconsCompreImg.fit";
  char nombreArMin_imag[] = "minCoefs_imag.txt";
  char nombreArCoef_imag[] = "coefs_imag.txt";
  char nombreArCoef_comp_imag[] = "coefs_comp_imag.txt";
  char nombreArMin_real[] = "minCoefs_real.txt";
  char nombreArCoef_real[] = "coefs_real.txt";
  char nombreArCoef_comp_real[] = "coefs_comp_real.txt";
  char nombreArInfoCompresion[] = "infoCompre.txt";
  char nombreArInfoTiemposEjecu[] = "infoTiemposEjecu.txt";


  // ############### CALCULO DE MU Y MV - CREACION DE DIRECTORIO SEGUNDARIO  ##############
  printf("...Comenzando calculo de MV...\n");
  clock_t tiempoCalculoMV;
  tiempoCalculoMV = clock();
  float* MV = calcularMV_Rect(v, delta_v, cantVisi, N, estrechezDeBorde, ancho, matrizDeUnos);
  tiempoCalculoMV = clock() - tiempoCalculoMV;
  float tiempoTotalCalculoMV = ((float)tiempoCalculoMV)/CLOCKS_PER_SEC;
  printf("Calculo de MV completado.\n");

  printf("...Comenzando calculo de MU...\n");
  clock_t tiempoCalculoMU;
  tiempoCalculoMU = clock();
  float* MU = calcularMV_Rect(u, delta_u, cantVisi, N, estrechezDeBorde, ancho, matrizDeUnos);
  tiempoCalculoMU = clock() - tiempoCalculoMU;
  float tiempoTotalCalculoMU = ((float)tiempoCalculoMU)/CLOCKS_PER_SEC;
  printf("Calculo de MU completado.\n");

  char* rutaADirecSec = (char*) malloc(strlen(nombreDirPrin)*strlen(nombreDirSec)*sizeof(char)+sizeof(char)*3);
  strcpy(rutaADirecSec, nombreDirPrin);
  strcat(rutaADirecSec, "/");
  strcat(rutaADirecSec, nombreDirSec);
  if(mkdir(rutaADirecSec, 0777) == -1)
  {
      printf("ERROR: No se pudo crear subdirectorio.");
      printf("PROGRAMA ABORTADO.\n");
      exit(0);
  }
  strcat(rutaADirecSec, "/");


  // ############### MINIMIZACION DE COEFS, PARTE IMAGINARIA  ##############
  char* nombreArchivoMin_imag = (char*) malloc(strlen(nombreDirPrin)*strlen(nombreDirSec)*strlen(nombreArMin_imag)*sizeof(char)+sizeof(char)*3);
  strcpy(nombreArchivoMin_imag, rutaADirecSec);
  strcat(nombreArchivoMin_imag, nombreArMin_imag);
  char* nombreArchivoCoefs_imag = (char*) malloc(strlen(nombreDirPrin)*strlen(nombreDirSec)*strlen(nombreArCoef_imag)*sizeof(char)+sizeof(char)*3);
  strcpy(nombreArchivoCoefs_imag, rutaADirecSec);
  strcat(nombreArchivoCoefs_imag, nombreArCoef_imag);
  printf("...Comenzando minimizacion de coeficientes parte imaginaria...\n");
  clock_t tiempoMinPartImag;
  tiempoMinPartImag = clock();
  float* MC_imag = minGradConjugado_MinCuadra_escritura(nombreArchivoMin_imag, nombreArchivoCoefs_imag, MV, MU, visi_parteImaginaria, w, cantVisi, N, matrizDeUnosTamN, maxIter, tol);
  tiempoMinPartImag = clock() - tiempoMinPartImag;
  float tiempoTotalMinPartImag = ((float)tiempoMinPartImag)/CLOCKS_PER_SEC;
  printf("Proceso de minimizacion de coeficientes parte imaginaria terminado.\n");
  free(nombreArchivoMin_imag);
  free(nombreArchivoCoefs_imag);


  // ############### MINIMIZACION DE COEFS, PARTE REAL  ##############
  char* nombreArchivoMin_real = (char*) malloc(strlen(nombreDirPrin)*strlen(nombreDirSec)*strlen(nombreArMin_real)*sizeof(char)+sizeof(char)*3);
  strcpy(nombreArchivoMin_real, rutaADirecSec);
  strcat(nombreArchivoMin_real, nombreArMin_real);
  char* nombreArchivoCoefs_real = (char*) malloc(strlen(nombreDirPrin)*strlen(nombreDirSec)*strlen(nombreArCoef_real)*sizeof(char)+sizeof(char)*3);
  strcpy(nombreArchivoCoefs_real, rutaADirecSec);
  strcat(nombreArchivoCoefs_real, nombreArCoef_real);
  printf("...Comenzando minimizacion de coeficientes parte real...\n");
  clock_t tiempoMinPartReal;
  tiempoMinPartReal = clock();
  float* MC_real = minGradConjugado_MinCuadra_escritura(nombreArchivoMin_real, nombreArchivoCoefs_real, MV, MU, visi_parteReal, w, cantVisi, N, matrizDeUnosTamN, maxIter, tol);
  tiempoMinPartReal = clock() - tiempoMinPartReal;
  float tiempoTotalMinPartReal = ((float)tiempoMinPartReal)/CLOCKS_PER_SEC;
  printf("Proceso de minimizacion de coeficientes parte real terminado.\n");
  free(nombreArchivoMin_real);
  free(nombreArchivoCoefs_real);


  // ############### CALCULO NIVEL DE INFORMACION ##############
  clock_t tiempoInfo;
  tiempoInfo = clock();
  float* medidasDeInfo = calInfoFisherDiag(MV, cantVisi, N, MU, w);
  tiempoInfo = clock() - tiempoInfo;
  float tiempoTotalInfo = ((float)tiempoInfo)/CLOCKS_PER_SEC;
  hipFree(MU);
  hipFree(MV);


  // ############### RECONSTRUCCION DEL PLANO GRILLEADO Y ALMACENAMIENTO DE LA RECONSTRUCCION DE LA IMAGEN ##############
  char* nombreArchivoReconsImg = (char*) malloc(strlen(nombreDirPrin)*strlen(nombreDirSec)*strlen(nombreArReconsImg)*sizeof(char)+sizeof(char)*3);
  strcpy(nombreArchivoReconsImg, rutaADirecSec);
  strcat(nombreArchivoReconsImg, nombreArReconsImg);
  clock_t tiempoCalculoMV_AF;
  tiempoCalculoMV_AF = clock();
  float* MV_AF = calcularMV_Rect_estFourier(ancho, N, delta_v, matrizDeUnos, estrechezDeBorde, matrizDeUnosTamN);
  tiempoCalculoMV_AF = clock() - tiempoCalculoMV_AF;
  float tiempoTotalCalculoMV_AF = ((float)tiempoCalculoMV_AF)/CLOCKS_PER_SEC;
  clock_t tiempoCalculoMU_AF;
  tiempoCalculoMU_AF = clock();
  float* MU_AF = calcularMV_Rect_estFourier(ancho, N, delta_u, matrizDeUnos, estrechezDeBorde, matrizDeUnosTamN);
  tiempoCalculoMU_AF = clock() - tiempoCalculoMU_AF;
  float tiempoTotalCalculoMU_AF = ((float)tiempoCalculoMU_AF)/CLOCKS_PER_SEC;
  clock_t tiempoReconsFourierPartImag;
  tiempoReconsFourierPartImag = clock();
  float* estimacionFourier_ParteImag = estimacionDePlanoDeFourier(MV_AF, N, N, MC_imag, N, N, MU_AF);
  tiempoReconsFourierPartImag = clock() - tiempoReconsFourierPartImag;
  float tiempoTotalReconsFourierPartImag = ((float)tiempoReconsFourierPartImag)/CLOCKS_PER_SEC;
  clock_t tiempoReconsFourierPartReal;
  tiempoReconsFourierPartReal = clock();
  float* estimacionFourier_ParteReal = estimacionDePlanoDeFourier(MV_AF, N, N, MC_real, N, N, MU_AF);
  tiempoReconsFourierPartReal = clock() - tiempoReconsFourierPartReal;
  float tiempoTotalReconsFourierPartReal = ((float)tiempoReconsFourierPartReal)/CLOCKS_PER_SEC;
  clock_t tiempoReconsTransInver;
  tiempoReconsTransInver = clock();
  escribirTransformadaInversaFourier2D(estimacionFourier_ParteImag, estimacionFourier_ParteReal, N, nombreArchivoReconsImg);
  tiempoReconsTransInver = clock() - tiempoReconsTransInver;
  float tiempoTotalReconsTransInver = ((float)tiempoReconsTransInver)/CLOCKS_PER_SEC;
  hipFree(estimacionFourier_ParteImag);
  hipFree(estimacionFourier_ParteReal);
  free(nombreArchivoReconsImg);


  // ############### CALCULO DE GRADO DE COMPRESION ##############
  char* rutaADirecTer = (char*) malloc(strlen(rutaADirecSec)*strlen(nombreDirTer)*sizeof(char)+sizeof(char)*3);
  strcpy(rutaADirecTer, rutaADirecSec);
  strcat(rutaADirecTer, "/");
  strcat(rutaADirecTer, nombreDirTer);
  if(mkdir(rutaADirecTer, 0777) == -1)
  {
    printf("ERROR: No se pudo crear subdirectorio.\n");
    printf("PROGRAMA ABORTADO.\n");
    exit(0);
  }
  strcat(rutaADirecTer, "/");
  clock_t tiempoReconsFourierPartImagComp;
  clock_t tiempoReconsFourierPartRealComp;
  clock_t tiempoReconsTransInverComp;
  printf("...Comenzando calculo de compresiones...\n");
  clock_t tiempoCompresion;
  tiempoCompresion = clock();
  int cantCoefs = calPSNRDeDistintasCompresiones(inicioPorcenCompre, terminoPorcenCompre, cantPorcen, rutaADirecSec, rutaADirecTer, nombreArReconsCompreImg, MC_imag, MC_real, MV_AF, MU_AF, N, &tiempoReconsFourierPartImagComp, &tiempoReconsFourierPartRealComp, &tiempoReconsTransInverComp);
  tiempoCompresion = clock() - tiempoCompresion;
  float tiempoTotalCompresion = ((float)tiempoCompresion)/CLOCKS_PER_SEC;
  printf("Proceso de calculo de compresiones terminado.\n");
  free(rutaADirecTer);
  char* nombreArchivoInfoComp = (char*) malloc(strlen(nombreDirPrin)*strlen(nombreArInfoCompresion)*sizeof(char)+sizeof(char)*2);
  strcpy(nombreArchivoInfoComp, nombreDirPrin);
  strcat(nombreArchivoInfoComp, "/");
  strcat(nombreArchivoInfoComp, nombreArInfoCompresion);
  FILE* archivo = fopen(nombreArchivoInfoComp, "a");
  float nivelDeCompresion = 1.0 - cantCoefs * 1.0 / N*N;
  fprintf(archivo, "%d %.12f %.12e %.12e %.12f %.12d\n", iterActual, ancho, medidasDeInfo[0], medidasDeInfo[1], nivelDeCompresion, cantCoefs);
  fclose(archivo);
  free(nombreArchivoInfoComp);
  free(medidasDeInfo);

  hipFree(MC_real);
  hipFree(MC_imag);
  hipFree(MU_AF);
  hipFree(MV_AF);
  float tiempoTotalReconsFourierPartImagComp = ((float)tiempoReconsFourierPartImagComp)/CLOCKS_PER_SEC;
  float tiempoTotalReconsFourierPartRealComp = ((float)tiempoReconsFourierPartRealComp)/CLOCKS_PER_SEC;
  float tiempoTotalReconsTransInverComp = ((float)tiempoReconsTransInverComp)/CLOCKS_PER_SEC;

  // ############### ESCRITURA DE ARCHIVO CON TIEMPOS DE EJECUCION ##############
  char* nombreArchivoInfoTiemposEjecu = (char*) malloc(strlen(nombreDirPrin)*strlen(nombreArInfoTiemposEjecu)*sizeof(char)+sizeof(char)*2);
  strcpy(nombreArchivoInfoTiemposEjecu, nombreDirPrin);
  strcat(nombreArchivoInfoTiemposEjecu, "/");
  strcat(nombreArchivoInfoTiemposEjecu, nombreArInfoTiemposEjecu);
  FILE* archivoInfoTiemposEjecu = fopen(nombreArchivoInfoTiemposEjecu, "a");
  fprintf(archivoInfoTiemposEjecu, "%d %.12f %.12e %.12e %.12e %.12e %.12e %.12e %.12e %.12e %.12e %.12e %.12e %.12e %.12e %.12e\n", iterActual, ancho, tiempoTotalCalculoMV, tiempoTotalCalculoMU, tiempoTotalMinPartImag, tiempoTotalMinPartReal, tiempoTotalInfo, tiempoTotalCompresion, tiempoTotalCalculoMV_AF, tiempoTotalCalculoMU_AF, tiempoTotalReconsFourierPartImag, tiempoTotalReconsFourierPartReal, tiempoTotalReconsTransInver, tiempoTotalReconsFourierPartImagComp, tiempoTotalReconsFourierPartRealComp, tiempoTotalReconsTransInverComp);
  fclose(archivoInfoTiemposEjecu);

  free(rutaADirecSec);
}

double funcOptiInfo_Traza_Rect(double ancho, void* params)
{
  struct parametros_BaseRect* ps = (struct parametros_BaseRect*) params;
  float* MV = calcularMV_Rect(ps->v, ps->delta_v, ps->cantVisi, ps->N, ps->estrechezDeBorde, ancho, ps->matrizDeUnos);
  float* MU = calcularMV_Rect(ps->u, ps->delta_u, ps->cantVisi, ps->N, ps->estrechezDeBorde, ancho, ps->matrizDeUnos);
  float* medidasDeInfo = calInfoFisherDiag(MV, ps->cantVisi, ps->N, MU, ps->w);
  float medidaSumaDeLaDiagonal = medidasDeInfo[0];
  free(medidasDeInfo);
  hipFree(MV);
  hipFree(MU);
  return -1 * medidaSumaDeLaDiagonal;
}

double funcOptiInfo_Traza_Normal(double ancho, void* params)
{
  struct parametros_BaseNormal* ps = (struct parametros_BaseNormal*) params;
  float* MV = calcularMV_Normal(ps->v, ps->delta_v, ps->cantVisi, ps->N, ancho);
  float* MU = calcularMV_Normal(ps->u, ps->delta_u, ps->cantVisi, ps->N, ancho);
  float* medidasDeInfo = calInfoFisherDiag(MV, ps->cantVisi, ps->N, MU, ps->w);
  float medidaSumaDeLaDiagonal = medidasDeInfo[0];
  free(medidasDeInfo);
  hipFree(MV);
  hipFree(MU);
  return -1 * medidaSumaDeLaDiagonal;
}

double goldenMin_BaseRect(float* u, float* v, float* w, float delta_u, float delta_v, float* matrizDeUnos, long cantVisi, long N, float estrechezDeBorde)
{
  int status;
  int iter = 0, max_iter = 100;
  const gsl_min_fminimizer_type *T;
  gsl_min_fminimizer *s;
  gsl_function F;
  parametros_BaseRect actual;
  actual.u = u;
  actual.v = v;
  actual.w = w;
  actual.delta_u = delta_u;
  actual.delta_v = delta_v;
  actual.matrizDeUnos = matrizDeUnos;
  actual.cantVisi = cantVisi;
  actual.N = N;
  actual.estrechezDeBorde = estrechezDeBorde;
  double m;
  double a = 1.0 * actual.delta_u, b = 5.0 * actual.delta_u;
  F.function = &funcOptiInfo_Traza_Rect;
  void* punteroVoidAActual = &actual;
  F.params = punteroVoidAActual;

  T = gsl_min_fminimizer_quad_golden;
  s = gsl_min_fminimizer_alloc (T);
  gsl_set_error_handler_off();

  m = 1.0 * actual.delta_u;
  int status_interval = gsl_min_fminimizer_set (s, &F, m, a, b);


  while(status_interval)
  {
    m += 0.001 * actual.delta_u;
    printf("m ahora es %f\n", m/actual.delta_u);
    status_interval = gsl_min_fminimizer_set (s, &F, m, a, b);
  }

  printf ("using %s method\n",
          gsl_min_fminimizer_name (s));

  printf ("%5s [%9s, %9s] %9s\n",
          "iter", "lower", "upper", "min");

  printf ("%5d [%.7f, %.7f] %.7f\n",
          iter, a, b, m);

  do
    {
      iter++;
      status = gsl_min_fminimizer_iterate (s);

      m = gsl_min_fminimizer_x_minimum (s);
      a = gsl_min_fminimizer_x_lower (s);
      b = gsl_min_fminimizer_x_upper (s);

      status = gsl_min_test_interval (a, b, 0.01, 0.01);

      if (status == GSL_SUCCESS)
        printf ("Converged:\n");

      printf ("%5d [%.7f, %.7f] "
              "%.7f\n",
              iter, a/delta_u, b/delta_u, m/delta_u);
    }
  while (status == GSL_CONTINUE && iter < max_iter);

  gsl_min_fminimizer_free (s);
  return m;
}

double goldenMin_BaseNormal(float* u, float* v, float* w, float delta_u, float delta_v, long cantVisi, long N)
{
  int status;
  int iter = 0, max_iter = 100;
  const gsl_min_fminimizer_type *T;
  gsl_min_fminimizer *s;
  gsl_function F;
  parametros_BaseNormal actual;
  actual.u = u;
  actual.v = v;
  actual.w = w;
  actual.delta_u = delta_u;
  actual.delta_v = delta_v;
  actual.cantVisi = cantVisi;
  actual.N = N;
  double m = 1.5 * actual.delta_u, m_expected = M_PI;
  double a = 1.0 * actual.delta_u, b = 5.0 * actual.delta_u;
  F.function = &funcOptiInfo_Traza_Normal;
  void* punteroVoidAActual = &actual;
  F.params = punteroVoidAActual;

  T = gsl_min_fminimizer_quad_golden;
  s = gsl_min_fminimizer_alloc (T);
  gsl_set_error_handler_off();

  m = 1.0 * actual.delta_u;
  int status_interval = gsl_min_fminimizer_set (s, &F, m, a, b);


  while(status_interval)
  {
    m += 0.001 * actual.delta_u;
    printf("m ahora es %f\n", m/actual.delta_u);
    status_interval = gsl_min_fminimizer_set (s, &F, m, a, b);
  }

  printf ("using %s method\n",
          gsl_min_fminimizer_name (s));

  printf ("%5s [%9s, %9s] %9s\n",
          "iter", "lower", "upper", "min");

  printf ("%5d [%.7f, %.7f] %.7f\n",
          iter, a, b, m);

  do
    {
      iter++;
      status = gsl_min_fminimizer_iterate (s);

      m = gsl_min_fminimizer_x_minimum (s);
      a = gsl_min_fminimizer_x_lower (s);
      b = gsl_min_fminimizer_x_upper (s);

      status
        = gsl_min_test_interval (a, b, 0.001, 0.0);

      if (status == GSL_SUCCESS)
        printf ("Converged:\n");

      printf ("%5d [%.7f, %.7f] "
              "%.7f\n",
              iter, a/delta_u, b/delta_u,m/delta_u);
    }
  while (status == GSL_CONTINUE && iter < max_iter);

  gsl_min_fminimizer_free (s);
  return m;
}

void lecturaDeTXT(char nombreArchivo[], float* frecuencia, float* u, float* v, float* w, float* visi_parteImaginaria, float* visi_parteReal, long cantVisi)
{
  long contador = 0;
  FILE *fp;
  size_t len = 0;
  char *line = NULL;
  ssize_t read;
  float c_constant = 2.99792458E8;
  fp = fopen(nombreArchivo, "r");
  if (fp == NULL)
  {
      printf("No se pudo abrir el archivo %s",nombreArchivo);
      exit(0);
  }
  while ((read = getline(&line, &len, fp)) != -1)
  {
    *frecuencia = atof(strtok(line, " "));
    visi_parteReal[contador] = atof(strtok(NULL, " "));
    visi_parteImaginaria[contador] = atof(strtok(NULL, " "));
    u[contador] = atof(strtok(NULL, " ")) * (*frecuencia)/c_constant;
    v[contador] = atof(strtok(NULL, " ")) * (*frecuencia)/c_constant;
    w[contador] = atof(strtok(NULL, " "));
    contador++;
    if(contador == cantVisi)
      break;
	}
  free(line);
  fclose(fp);
}

void lectCantVisi(char nombreArchivo[], long* cantVisi)
{
  long contador = 0;
  FILE *fp;
  size_t len = 0;
  char *line = NULL;
  ssize_t read;
  char* nombreNuevoTXT = (char*) malloc(strlen(nombreArchivo)*sizeof(char)+sizeof(char)*20);
  strcpy(nombreNuevoTXT, nombreArchivo);
  strcat(nombreNuevoTXT, "cantvisi.txt");
  fp = fopen(nombreNuevoTXT, "r");
  if (fp == NULL)
  {
      printf("No se pudo abrir el archivo %s",nombreArchivo);
      exit(0);
  }
  read = getline(&line, &len, fp);
  printf("Se han leido %s visibilidades.\n", line);
  *cantVisi = atoi(line);
  free(line);
  free(nombreNuevoTXT);
  fclose(fp);
}

void lectDeTXTcreadoDesdeMS(char nombreArchivo[], float* u, float* v, float* w, float* visi_parteImaginaria, float* visi_parteReal)
{
  long contador = 0;
  FILE *fp;
  size_t len = 0;
  char *line = NULL;
  ssize_t read;
  char* nombreNuevoTXT = (char*) malloc(strlen(nombreArchivo)*sizeof(char)+sizeof(char)*5);
  strcpy(nombreNuevoTXT, nombreArchivo);
  strcat(nombreNuevoTXT, ".txt");
  fp = fopen(nombreNuevoTXT, "r");
  if (fp == NULL)
  {
      printf("No se pudo abrir el archivo %s",nombreArchivo);
      exit(0);
  }
  while ((read = getline(&line, &len, fp)) != -1)
  {
    visi_parteReal[contador] = atof(strtok(line, " "));
    visi_parteImaginaria[contador] = atof(strtok(NULL, " "));
    u[contador] = atof(strtok(NULL, " "));
    v[contador] = atof(strtok(NULL, " "));
    w[contador] = atof(strtok(NULL, " "));
    contador++;
	}
  printf("El contador es %ld\n", contador);
  free(line);
  free(nombreNuevoTXT);
  fclose(fp);
}

void lectDeTXTcreadoDesdeMSConLimite(char nombreArchivo[], float* u, float* v, float* w, float* visi_parteImaginaria, float* visi_parteReal, long inicio, long fin, long cantVisi)
{
  long contador = 0;
  long contadorIte = 0;
  FILE *fp;
  size_t len = 0;
  char *line = NULL;
  ssize_t read;
  char* nombreNuevoTXT = (char*) malloc(strlen(nombreArchivo)*sizeof(char)+sizeof(char)*5);
  strcpy(nombreNuevoTXT, nombreArchivo);
  strcat(nombreNuevoTXT, ".txt");
  fp = fopen(nombreNuevoTXT, "r");
  printf("Nombre nuevo es %s\n", nombreNuevoTXT);
  if (fp == NULL)
  {
      printf("No se pudo abrir el archivo %s",nombreArchivo);
      exit(0);
  }
  while ((read = getline(&line, &len, fp)) != -1)
  {
    if (contadorIte >= inicio)
    {
      visi_parteReal[contador] = atof(strtok(line, " "));
      visi_parteImaginaria[contador] = atof(strtok(NULL, " "));
      u[contador] = atof(strtok(NULL, " "));
      v[contador] = atof(strtok(NULL, " "));
      w[contador] = atof(strtok(NULL, " "));
      contador++;
    }
    contadorIte++;
    if(contadorIte >= fin)
      break;
	}
  printf("El contador es %ld\n", contador);
  free(line);
  free(nombreNuevoTXT);
  fclose(fp);
}

void escrituraDeArchivoConParametros_Normal(char nombreArchivoPara[], char nombreArchivo[], char nombreDirPrin[], int cantVisi, int N, int maxIter, float tolGrad)
{
  time_t t = time(NULL);
  struct tm tm = *localtime(&t);
  FILE* archivoDePara = fopen(nombreArchivoPara, "w");
  fprintf(archivoDePara, "Programa inicio su ejecucion con fecha: %d-%d-%d %d:%d:%d\n", tm.tm_year + 1900, tm.tm_mon + 1,tm.tm_mday, tm.tm_hour, tm.tm_min, tm.tm_sec);
  fprintf(archivoDePara, "Compresion con base normal utilizando informacion del archivo %s cuyos parametros de ejecucion fueron:\n", nombreArchivo);
  fprintf(archivoDePara, "Cantidad de visibilidades(cantVisi): %d\n", cantVisi);
  fprintf(archivoDePara, "Cantidad de Coefs(N x N): %d x %d = %d\n", N, N, N*N);
  fprintf(archivoDePara, "Maximo de iteraciones impuesto para la minimizacion de coeficientes(maxIter): %d\n", maxIter);
  fprintf(archivoDePara, "Grado de tolerancia a la minimizacion de los coefs(tolGrad): %.12e\n", tolGrad);
  fclose(archivoDePara);
}

void escrituraDeArchivoConParametros_Rect(char nombreArchivoPara[], char nombreArchivo[], char nombreDirPrin[], long cantVisi, long N, int maxIter, float tolGrad, float estrechezDeBorde)
{
  time_t t = time(NULL);
  struct tm tm = *localtime(&t);
  FILE* archivoDePara = fopen(nombreArchivoPara, "w");
  fprintf(archivoDePara, "Programa inicio su ejecucion con fecha: %d-%d-%d %d:%d:%d\n", tm.tm_year + 1900, tm.tm_mon + 1,tm.tm_mday, tm.tm_hour, tm.tm_min, tm.tm_sec);
  fprintf(archivoDePara, "Compresion con base rectangular utilizando informacion del archivo %s cuyos parametros de ejecucion fueron:\n", nombreArchivo);
  fprintf(archivoDePara, "Estrechez de borde: %f\n", estrechezDeBorde);
  fprintf(archivoDePara, "Cantidad de visibilidades(cantVisi): %ld\n", cantVisi);
  fprintf(archivoDePara, "Cantidad de Coefs(N x N): %ld x %ld = %ld\n", N, N, N*N);
  fprintf(archivoDePara, "Maximo de iteraciones impuesto para la minimizacion de coeficientes(maxIter): %d\n", maxIter);
  fprintf(archivoDePara, "Grado de tolerancia a la minimizacion de los coefs(tolGrad): %.12e\n", tolGrad);
  fclose(archivoDePara);
}

void calculoDeInfoCompre_BaseNormal(char nombreArchivo[], int maxIter, float tolGrad, float tolGolden, float* u, float* v, float* w, float* visi_parteImaginaria, float* visi_parteReal, float delta_u, float delta_v, long cantVisi, long N, float cotaEnergia, char nombreDirPrin[], char nombreDirSec[], int cantParamEvaInfo, float inicioIntervalo, float finIntervalo, float* matrizDeUnosEstFourier, float estrechezDeBorde)
{
  float inicioIntervaloEscalado = inicioIntervalo * delta_u;
  float finIntervaloEscalado = finIntervalo * delta_u;
  char nombreArPara[] = "parametrosEjecucion.txt";
  if(cotaEnergia > 1.0)
  {
      printf("ERROR: La cota de energia debe estar expresado en decimales, no en porcentajes.\n");
      printf("PROGRAMA ABORTADO.\n");
      exit(0);
  }
  int cotaEnergiaInt = cotaEnergia * 100;
  char* cotaEnergiaString = numAString(&cotaEnergiaInt);
  sprintf(cotaEnergiaString, "%d", cotaEnergiaInt);
  strcat(nombreDirPrin, cotaEnergiaString);
  if(mkdir(nombreDirPrin, 0777) == -1)
  {
      printf("ERROR: El directorio EXISTE, PELIGRO DE SOBREESCRITURA, por favor eliga otro nombre de directorio.\n");
      printf("PROGRAMA ABORTADO.\n");
      exit(0);
  }
  else
      printf("Directorio creado.\n");
  char* nombreArchivoPara = (char*) malloc(strlen(nombreDirPrin)*strlen(nombreArPara)*sizeof(char)+sizeof(char)*3);
  strcpy(nombreArchivoPara, nombreDirPrin);
  strcat(nombreArchivoPara, "/");
  strcat(nombreArchivoPara, nombreArPara);
  escrituraDeArchivoConParametros_Rect(nombreArchivoPara, nombreArchivo, nombreDirPrin, cantVisi, N, maxIter, tolGrad, estrechezDeBorde);
  free(nombreArchivoPara);

  // goldenMin_BaseNormal(u, v, w, delta_u, delta_v, cantVisi, N);
  // printf("El optimo esta en %.12f\n", optimo);

  // printf("inicio del intervalo es %.12f y el fin del intervalo es %.12f\n", inicioIntervalo, finIntervalo);
  // float* paramEvaInfo = linspace(inicioIntervaloEscalado, finIntervaloEscalado, cantParamEvaInfo);
  // int i = 0;
  // // for(int i=0; i<cantParamEvaInfo; i++)
  // // {
  //   char* numComoString = numAString(&i);
  //   sprintf(numComoString, "%d", i);
  //   char* nombreDirSecCopia = (char*) malloc(sizeof(char)*strlen(nombreDirSec)*strlen(numComoString));
  //   strcpy(nombreDirSecCopia, nombreDirSec);
  //   strcat(nombreDirSecCopia, numComoString);
  //   calCompSegunAncho_Normal_escritura(nombreDirPrin, nombreDirSecCopia, paramEvaInfo[i], cotaEnergia, i, maxIter, tolGrad, u, v, w, visi_parteImaginaria, visi_parteReal, delta_u, delta_v, cantVisi, N, matrizDeUnosEstFourier, estrechezDeBorde);
  //   free(numComoString);
  //   free(nombreDirSecCopia);
  // }
}

void calculoDeInfoCompre_BaseRect(char nombreArchivo[], int maxIter, float tolGrad, float tolGolden, float* u, float* v, float* w, float* visi_parteImaginaria, float* visi_parteReal, float delta_u, float delta_v, float* matrizDeUnos, long cantVisi, long N, float cotaEnergia, char nombreDirPrin[], char nombreDirSec[], char nombreDirTer[], int cantParamEvaInfo, float inicioIntervalo, float finIntervalo, float* matrizDeUnosEstFourier, float estrechezDeBorde)
{
  float inicioIntervaloEscalado = inicioIntervalo * delta_u;
  float finIntervaloEscalado = finIntervalo * delta_u;
  char nombreArPara[] = "parametrosEjecucion.txt";
  if(cotaEnergia > 1.0)
  {
      printf("ERROR: La cota de energia debe estar expresado en decimales, no en porcentajes.\n");
      printf("PROGRAMA ABORTADO.\n");
      exit(0);
  }
  int cotaEnergiaInt = cotaEnergia * 100;
  char* cotaEnergiaString = numAString(&cotaEnergiaInt);
  sprintf(cotaEnergiaString, "%d", cotaEnergiaInt);
  // strcat(nombreDirPrin, cotaEnergiaString);
  if(mkdir(nombreDirPrin, 0777) == -1)
  {
      printf("ERROR: El directorio EXISTE, PELIGRO DE SOBREESCRITURA, por favor eliga otro nombre de directorio.\n");
      printf("PROGRAMA ABORTADO.\n");
      exit(0);
  }
  else
      printf("Directorio creado.\n");
  char* nombreArchivoPara = (char*) malloc(strlen(nombreDirPrin)*strlen(nombreArPara)*sizeof(char)+sizeof(char)*3);
  strcpy(nombreArchivoPara, nombreDirPrin);
  strcat(nombreArchivoPara, "/");
  strcat(nombreArchivoPara, nombreArPara);
  escrituraDeArchivoConParametros_Rect(nombreArchivoPara, nombreArchivo, nombreDirPrin, cantVisi, N, maxIter, tolGrad, estrechezDeBorde);
  free(nombreArchivoPara);

  // float optimo = goldenMin_BaseRect(u, v, w, delta_u, delta_v, matrizDeUnos, cantVisi, N, estrechezDeBorde);
  // printf("El optimo esta en %.12f\n", optimo);

  float* paramEvaInfo = linspace(inicioIntervaloEscalado, finIntervaloEscalado, cantParamEvaInfo);
  int i = 0;
  // for(int i=0; i<cantParamEvaInfo; i++)
  // {
    char* numComoString = numAString(&i);
    sprintf(numComoString, "%d", i);
    char* nombreDirSecCopia = (char*) malloc(sizeof(char)*strlen(nombreDirSec)*strlen(numComoString));
    strcpy(nombreDirSecCopia, nombreDirSec);
    strcat(nombreDirSecCopia, numComoString);
    calCompSegunAncho_Rect_escritura(nombreDirPrin, nombreDirSecCopia, nombreDirTer, paramEvaInfo[i], cotaEnergia, i, maxIter, tolGrad, u, v, w, visi_parteImaginaria, visi_parteReal, delta_u, delta_v, matrizDeUnos, cantVisi, N, matrizDeUnosEstFourier, estrechezDeBorde);
    free(numComoString);
    free(nombreDirSecCopia);
  // }
}

void calImagenesADistintasCompresiones_Rect(float inicioIntervalo, float finIntervalo, int cantParamEvaInfo, char nombreDirPrin[], float ancho, int maxIter, float tol, float* u, float* v, float* w, float* visi_parteImaginaria, float* visi_parteReal, float delta_u, float delta_v, float* matrizDeUnos, long cantVisi, long N, float* matrizDeUnosTamN, float estrechezDeBorde)
{

  if(mkdir(nombreDirPrin, 0777) == -1)
  {
      printf("ERROR: El directorio EXISTE, PELIGRO DE SOBREESCRITURA, por favor eliga otro nombre de directorio.\n");
      printf("PROGRAMA ABORTADO.\n");
      exit(0);
  }
  else
      printf("Directorio creado.\n");
  char nombreArReconsCompreImg[] = "reconsCompreImg";
  float* paramEvaInfo = linspace(inicioIntervalo/100.0, finIntervalo/100.0, cantParamEvaInfo);


  // ############### CALCULO DE MU Y MV - CREACION DE DIRECTORIO SEGUNDARIO  ##############
  printf("...Comenzando calculo de MV...\n");
  float* MV = calcularMV_Rect(v, delta_v, cantVisi, N, estrechezDeBorde, ancho, matrizDeUnos);
  printf("Calculo de MV completado.\n");

  printf("...Comenzando calculo de MU...\n");
  float* MU = calcularMV_Rect(u, delta_u, cantVisi, N, estrechezDeBorde, ancho, matrizDeUnos);
  printf("Calculo de MU completado.\n");


  // ############### MINIMIZACION DE COEFS, PARTE IMAGINARIA  ##############
  printf("...Comenzando minimizacion de coeficientes parte imaginaria...\n");
  float* MC_imag = minGradConjugado_MinCuadra(MV, MU, visi_parteImaginaria, w, cantVisi, N, matrizDeUnosTamN, maxIter, tol);
  printf("Proceso de minimizacion de coeficientes parte imaginaria terminado.\n");


  // ############### MINIMIZACION DE COEFS, PARTE REAL  ##############
  printf("...Comenzando minimizacion de coeficientes parte real...\n");
  float* MC_real = minGradConjugado_MinCuadra(MV, MU, visi_parteReal, w, cantVisi, N, matrizDeUnosTamN, maxIter, tol);
  printf("Proceso de minimizacion de coeficientes parte real terminado.\n");


  float* MV_AF = calcularMV_Rect_estFourier(ancho, N, delta_v, matrizDeUnos, estrechezDeBorde, matrizDeUnosTamN);
  float* MU_AF = calcularMV_Rect_estFourier(ancho, N, delta_u, matrizDeUnos, estrechezDeBorde, matrizDeUnosTamN);


  float* MC_comp_imag;
  hipMallocManaged(&MC_comp_imag,N*N*sizeof(float));
  hipMemset(MC_comp_imag, 0, N*N*sizeof(float));
  float* MC_comp_real;
  hipMallocManaged(&MC_comp_real,N*N*sizeof(float));
  hipMemset(MC_comp_real, 0, N*N*sizeof(float));

  long largo = N * N;
  float* MC_img_cuadrado;
  hipMallocManaged(&MC_img_cuadrado, N*N*sizeof(float));
  float* MC_modulo;
  hipMallocManaged(&MC_modulo, N*N*sizeof(float));
  hadamardProduct(MC_imag, N, N, MC_imag, MC_img_cuadrado);
  hadamardProduct(MC_real, N, N, MC_real, MC_modulo);
  combinacionLinealMatrices(1.0, MC_img_cuadrado, N, N, 1.0, MC_modulo);
  hipFree(MC_img_cuadrado);
  af::array MC_modulo_GPU(N*N, MC_modulo);
  hipFree(MC_modulo);
  af::array MC_modulo_indicesOrde_GPU(N*N);
  af::array MC_modulo_Orde_GPU(N*N);
  af::sort(MC_modulo_Orde_GPU, MC_modulo_indicesOrde_GPU, MC_modulo_GPU, 0, false);
  float total = af::sum<float>(MC_modulo_GPU);
  MC_modulo_Orde_GPU = MC_modulo_Orde_GPU/total;
  af::eval(MC_modulo_Orde_GPU);
  af::eval(MC_modulo_indicesOrde_GPU);
  af::sync();
  float* auxiliar_MC_modulo_Orde_GPU = MC_modulo_Orde_GPU.device<float>();
  float* auxiliar_MC_modulo_indicesOrde_GPU = MC_modulo_indicesOrde_GPU.device<float>();
  float* coefsNormalizados = (float*) malloc(largo*sizeof(float));
  hipMemcpy(coefsNormalizados, auxiliar_MC_modulo_Orde_GPU, N*N*sizeof(float), hipMemcpyDeviceToHost);
  int* MC_modulo_indicesOrde_CPU = (int*) malloc(largo*sizeof(int));
  hipMemcpy(MC_modulo_indicesOrde_CPU, auxiliar_MC_modulo_indicesOrde_GPU, N*N*sizeof(int), hipMemcpyDeviceToHost);
  MC_modulo_Orde_GPU.unlock();
  MC_modulo_GPU.unlock();
  MC_modulo_indicesOrde_GPU.unlock();

  long cantCoefsParaCota = 0;
  float sumador = 0.0;
  float* cantCoefsPorParametro = (float*) malloc(sizeof(float)*cantParamEvaInfo);
  float* cantidadPorcentualDeCoefs = linspace(1.0, largo, largo);
  combinacionLinealMatrices(0.0, cantidadPorcentualDeCoefs, largo, 1, 1.0/N, cantidadPorcentualDeCoefs);
  for(long j=0; j<cantParamEvaInfo; j++)
  {
    sumador = 0.0;
    cantCoefsParaCota = 0;
    for(long i=0; i<largo; i++)
    {
       sumador += coefsNormalizados[i];
       cantCoefsParaCota++;
       if(cantidadPorcentualDeCoefs[i] >= paramEvaInfo[j])
       {
         printf("Del %f%% solicitado, se ha tomado el mas cercano correspondiente al %f%% de coefs, lo que corresponde a un total de %ld coeficientes los cuales poseen el %f%% de la energia.\n", paramEvaInfo[j], cantidadPorcentualDeCoefs[i], cantCoefsParaCota, sumador);
         break;
       }
    }
    float* indicesATomar_CPU = (float*) malloc(cantCoefsParaCota*sizeof(float));
    for(int k=0; k<cantCoefsParaCota; k++)
    {
      indicesATomar_CPU[k] = MC_modulo_indicesOrde_CPU[k];
    }
    af::array indicesATomar_GPU(cantCoefsParaCota, indicesATomar_CPU);
    free(indicesATomar_CPU);
    af::array indRepComp = af::constant(0, largo);
    indRepComp(indicesATomar_GPU) = 1;
    indicesATomar_GPU.unlock();

    af::array MC_imag_GPU(N*N, MC_imag);
    af::array MC_real_GPU(N*N, MC_real);
    MC_imag_GPU = MC_imag_GPU * indRepComp;
    MC_real_GPU = MC_real_GPU * indRepComp;
    af::eval(MC_imag_GPU);
    af::eval(MC_real_GPU);
    af::sync();
    indRepComp.unlock();
    float* auxiliar_MC_imag_GPU = MC_imag_GPU.device<float>();
    float* auxiliar_MC_real_GPU = MC_real_GPU.device<float>();
    hipMemcpy(MC_comp_imag, auxiliar_MC_imag_GPU, N*N*sizeof(float), hipMemcpyDeviceToHost);
    MC_imag_GPU.unlock();
    hipMemcpy(MC_comp_real, auxiliar_MC_real_GPU, N*N*sizeof(float), hipMemcpyDeviceToHost);
    MC_real_GPU.unlock();
    float* estimacionFourier_compre_ParteImag = estimacionDePlanoDeFourier(MV_AF, N, N, MC_comp_imag, N, N, MU_AF);
    float* estimacionFourier_compre_ParteReal = estimacionDePlanoDeFourier(MV_AF, N, N, MC_comp_real, N, N, MU_AF);
    int numero = j+1;
    char* numComoString = numAString(&numero);
    sprintf(numComoString, "%d", numero);
    char* nombreArchivoReconsImgComp = (char*) malloc(sizeof(char)*strlen(nombreDirPrin)*strlen(numComoString)*strlen(nombreArReconsCompreImg)+sizeof(char)*7);
    strcpy(nombreArchivoReconsImgComp, nombreDirPrin);
    strcat(nombreArchivoReconsImgComp, "/");
    strcat(nombreArchivoReconsImgComp, nombreArReconsCompreImg);
    strcat(nombreArchivoReconsImgComp, "_");
    strcat(nombreArchivoReconsImgComp, numComoString);
    strcat(nombreArchivoReconsImgComp, ".fit");

    printf("%s\n", nombreArchivoReconsImgComp);

    escribirTransformadaInversaFourier2D(estimacionFourier_compre_ParteImag, estimacionFourier_compre_ParteReal, N, nombreArchivoReconsImgComp);
    hipFree(estimacionFourier_compre_ParteImag);
    hipFree(estimacionFourier_compre_ParteReal);
    free(numComoString);
    free(nombreArchivoReconsImgComp);
  }
  hipFree(MU_AF);
  hipFree(MV_AF);
  free(coefsNormalizados);
  free(MC_modulo_indicesOrde_CPU);
}

void filtroGaussiano()
{
  int largoVector = 100;
  float* porcenReal = (float*) malloc(sizeof(float)*largoVector);
  float* vector = (float*) malloc(sizeof(float)*largoVector);
  long contador = 0;
  FILE *fp;
  size_t len = 0;
  char *line = NULL;
  ssize_t read;
  fp = fopen("/home/rarmijo/psnr_hd142_rect.txt", "r");
  if (fp == NULL)
  {
      printf("No se pudo abrir el archivo");
      exit(0);
  }
  while ((read = getline(&line, &len, fp)) != -1)
  {
    porcenReal[largoVector-1-contador] = atof(strtok(line, " "));
    strtok(NULL, " ");
    vector[contador] = atof(strtok(NULL, " "));
    contador++;
	}
  printf("El contador es %ld\n", contador);
  free(line);
  fclose(fp);

  // for(int i=0; i<largoVector; i++)
  // {
  //   printf("%f\n", porcenReal[i]);
  // }
  // exit(-1);

  float* vectorFiltrado = (float*) calloc(largoVector, sizeof(float));
  gsl_vector* copiaVectorEnGSL = gsl_vector_alloc(largoVector);
  gsl_vector* vectorEnGSLFiltrado = gsl_vector_alloc(largoVector);
  for(int i=0; i<largoVector; i++)
  {
    gsl_vector_set(copiaVectorEnGSL, i, vector[largoVector-1-i]);
  }
  gsl_filter_gaussian_workspace* gauss_p = gsl_filter_gaussian_alloc(largoVector);
  gsl_filter_gaussian(GSL_FILTER_END_TRUNCATE, 1.0, 0, copiaVectorEnGSL, vectorEnGSLFiltrado, gauss_p);
  for(int i=0; i<largoVector; i++)
  {
    vectorFiltrado[i] = gsl_vector_get(copiaVectorEnGSL, i);
  }
  gsl_vector_free(copiaVectorEnGSL);
  gsl_vector_free(vectorEnGSLFiltrado);
  gsl_filter_gaussian_free(gauss_p);

  float* listaDeMetricas = (float*) malloc(sizeof(float)*largoVector);
  float* primeraRecta_subListaDeX = (float*) calloc(largoVector, sizeof(float));
  float* primeraRecta_subListaDeY = (float*) calloc(largoVector, sizeof(float));
  float* segundaRecta_subListaDeX = (float*) calloc(largoVector, sizeof(float));
  float* segundaRecta_subListaDeY = (float*) calloc(largoVector, sizeof(float));
  memcpy(segundaRecta_subListaDeX, porcenReal, sizeof(float)*largoVector);
  memcpy(segundaRecta_subListaDeY, vectorFiltrado, sizeof(float)*largoVector);
  primeraRecta_subListaDeX[0] = porcenReal[0];
  primeraRecta_subListaDeY[0] = vectorFiltrado[0];
  for(int i=1; i<largoVector-1; i++)
  {
      primeraRecta_subListaDeX[i] = porcenReal[i];
      primeraRecta_subListaDeY[i] = vectorFiltrado[i];
      float pendienteDePrimeraRecta = calPendiente(primeraRecta_subListaDeX, i+1, primeraRecta_subListaDeY);
      // printf("En la iteracion %d la pendienteDePrimeraRecta es %f\n", i, pendienteDePrimeraRecta);
      segundaRecta_subListaDeX[i-1] = 0.0;
      segundaRecta_subListaDeY[i-1] = 0.0;
      float pendienteDeSegundaRecta = calPendiente(&(segundaRecta_subListaDeX[i]), largoVector-i, &(segundaRecta_subListaDeY[i]));
      // printf("En la iteracion %d la pendienteDeSegundaRecta es %f\n", i, pendienteDeSegundaRecta);
      listaDeMetricas[i] = -1 * pendienteDeSegundaRecta/pendienteDePrimeraRecta;
      printf("%f\n", listaDeMetricas[i]);
  }
  free(primeraRecta_subListaDeX);
  free(primeraRecta_subListaDeY);
  free(segundaRecta_subListaDeX);
  free(segundaRecta_subListaDeY);
}

int main()
{
  // PARAMETROS GENERALES
  long cantVisi = 15034;
  long inicio = 0;
  long fin = 15034;

  // long cantVisi = 30000;
  // long inicio = 0;
  // long fin = 30000;

  int N = 512;
  // long N = 1600; //HLTau_B6cont.calavg.tav300s
  int maxIter = 100;

  float tolGrad = 1E-12;

  float delta_x = 0.02;
  // float delta_x = 0.005; //HLTau_B6cont.calavg.tav300s
  // float delta_x = 0.03; //co65
  float delta_x_rad = (delta_x * M_PI)/648000.0;
  float delta_u = 1.0/(N*delta_x_rad);
  float delta_v = 1.0/(N*delta_x_rad);

  //PARAMETROS PARTICULARES DE BASE RECT
  float estrechezDeBorde = 1000.0;

  // float frecuencia;
  // float *u, *v, *w, *visi_parteImaginaria, *visi_parteReal;
  // hipMallocManaged(&u, cantVisi*sizeof(float));
  // hipMallocManaged(&v, cantVisi*sizeof(float));
  // hipMallocManaged(&w, cantVisi*sizeof(float));
  // hipMallocManaged(&visi_parteImaginaria, cantVisi*sizeof(float));
  // hipMallocManaged(&visi_parteReal, cantVisi*sizeof(float));
  // char nombreArchivo[] = "hd142_b9cont_self_tav.0.0.txt";
  // lecturaDeTXT(nombreArchivo, &frecuencia, u, v, w, visi_parteImaginaria, visi_parteReal, cantVisi);

  // // ########### NOTEBOOK ##############
  // char nombreArchivo[] = "/home/yoyisaurio/Desktop/HLTau_B6cont.calavg.tav300s";
  // char comandoCasaconScript[] = "/home/yoyisaurio/casa-pipeline-release-5.6.2-2.el7/bin/casa -c /home/yoyisaurio/Desktop/proyecto/deMSaTXT.py";

  // // ########### PC-LAB ##############
  // char nombreArchivo[] = "/home/rarmijo/Desktop/proyecto/HLTau_B6cont.calavg.tav300s";
  // char comandoCasaconScript[] = "/home/rarmijo/casa-pipeline-release-5.6.2-2.el7/bin/casa -c ./deMSaTXT.py";

  // // ########### PC-LAB ##############
  // char nombreArchivo[] = "./co65.ms";
  // char comandoCasaconScript[] = "/home/rarmijo/casa-pipeline-release-5.6.2-2.el7/bin/casa -c ./deMSaTXT.py";

  // // ########### BEAM ##############
  // char nombreArchivo[] = "./HLTau_B6cont.calavg.tav300s";
  // char comandoCasaconScript[] = "casa -c ./deMSaTXT.py";

  // // ########### BEAM ##############
  // char nombreArchivo[] = "./FREQ78.ms";
  // char comandoCasaconScript[] = "casa -c ./deMSaTXT.py";

  // // // ########### BEAM ##############
  // char nombreArchivo[] = "./co65.ms";
  // char comandoCasaconScript[] = "casa -c ./deMSaTXT.py";

  // ########### BEAM ##############
  char nombreArchivo[] = "./hd142_b9cont_self_tav.ms";
  char comandoCasaconScript[] = "casa -c ./deMSaTXT.py";

  // // ########### BEAM ##############
  // char nombreArchivo[] = "/home/rarmijo/HLTau_Band6_CalibratedData/HLTau_B6cont.calavg";
  // char comandoCasaconScript[] = "casa -c ./deMSaTXT.py";

  // char* comandoScriptMSaTXT = (char*) malloc(strlen(comandoCasaconScript)*strlen(nombreArchivo)*sizeof(char)+sizeof(char)*3);
  // strcpy(comandoScriptMSaTXT, comandoCasaconScript);
  // strcat(comandoScriptMSaTXT, " ");
  // strcat(comandoScriptMSaTXT, nombreArchivo);
  // system(comandoScriptMSaTXT);
  // free(comandoScriptMSaTXT);


  lectCantVisi(nombreArchivo, &cantVisi);
  float *u, *v, *w, *visi_parteImaginaria, *visi_parteReal;
  hipMallocManaged(&u, cantVisi*sizeof(float));
  hipMallocManaged(&v, cantVisi*sizeof(float));
  hipMallocManaged(&w, cantVisi*sizeof(float));
  hipMallocManaged(&visi_parteImaginaria, cantVisi*sizeof(float));
  hipMallocManaged(&visi_parteReal, cantVisi*sizeof(float));
  lectDeTXTcreadoDesdeMS(nombreArchivo, u, v, w, visi_parteImaginaria, visi_parteReal);
  // lectDeTXTcreadoDesdeMSConLimite(nombreArchivo, u, v, w, visi_parteImaginaria, visi_parteReal, inicio, fin, cantVisi);

  float* matrizDeUnos, *matrizDeUnosEstFourier;
  hipMallocManaged(&matrizDeUnos, cantVisi*N*sizeof(float));
  for(long i=0; i<(cantVisi*N); i++)
  {
    matrizDeUnos[i] = 1.0;
  }
  hipMallocManaged(&matrizDeUnosEstFourier, N*sizeof(float));
  for(long i=0; i<N; i++)
  {
    matrizDeUnosEstFourier[i] = 1.0;
  }

  // float* rango = linspace(1.0 * delta_u, 5.0 * delta_u, 100);
  // for(int i=0; i<100; i++)
  // {
  //   float* MV = calcularMV_Rect(v, delta_v, cantVisi, N, estrechezDeBorde, rango[i], matrizDeUnos);
  //   float* MU = calcularMV_Rect(u, delta_u, cantVisi, N, estrechezDeBorde, rango[i], matrizDeUnos);
  //   float* medidasDeInfo = calInfoFisherDiag(MV, cantVisi, N, MU, w);
  //   hipFree(MU);
  //   hipFree(MV);
  //   float medidaSumaDeLaDiagonal = medidasDeInfo[0];
  //   free(medidasDeInfo);
  //   float info = -1 * medidaSumaDeLaDiagonal;
  //   printf("%.12e\n", info);
  // }


  // goldenMin(u, v, w, delta_u, delta_v, matrizDeUnos, cantVisi, N, estrechezDeBorde);
  //
  // // filtroGaussiano();
  // exit(-1);


  // // double ancho = delta_u;
  // //
  // // // float* MV = calcularMV_Normal(v, delta_v, cantVisi, N, ancho);
  // // float* MV = calcularMV_Rect(v, delta_v, cantVisi, N, estrechezDeBorde, ancho, matrizDeUnos);
  // // // float* MU = calcularMV_Normal(u, delta_u, cantVisi, N, ancho);
  // // float* MU = calcularMV_Rect(u, delta_u, cantVisi, N, estrechezDeBorde, ancho, matrizDeUnos);
  // // float* medidasDeInfo = calInfoFisherDiag(MV, cantVisi, N, MU, w);
  // // float medidaSumaDeLaDiagonal = medidasDeInfo[0];
  // // printf("%.12e\n", medidaSumaDeLaDiagonal);


  float cotaEnergia = 0.99;
  // char nombreDirPrin[] = "float_calCompresion_baseNormal_cota";
  char nombreDirPrin[] = "experi_hd142_solo80";
  char nombreDirSec[] = "ite";
  char nombreDirTer[] = "compresiones";
  char nombreArchivoTiempo[] = "tiempo.txt";
  int cantParamEvaInfo = 80;
  // float inicioIntervalo = 0.8;
  float inicioIntervalo = 1.0;
  float finIntervalo = 3.0;
  float tolGolden = 1E-12;
  int iterActual = 0;
  clock_t t;
  t = clock();
  // calculoDeInfoCompre_BaseNormal(nombreArchivo, maxIter, tolGrad, tolGolden, u, v, w, visi_parteImaginaria, visi_parteReal, delta_u, delta_v, cantVisi, N, cotaEnergia, nombreDirPrin, nombreDirSec, nombreDirTer, cantParamEvaInfo, inicioIntervalo, finIntervalo, matrizDeUnosEstFourier, estrechezDeBorde);
  calculoDeInfoCompre_BaseRect(nombreArchivo, maxIter, tolGrad, tolGolden, u, v, w, visi_parteImaginaria, visi_parteReal, delta_u, delta_v, matrizDeUnos, cantVisi, N, cotaEnergia, nombreDirPrin, nombreDirSec, nombreDirTer, cantParamEvaInfo, inicioIntervalo, finIntervalo, matrizDeUnosEstFourier, estrechezDeBorde);
  t = clock() - t;
  float time_taken = ((float)t)/CLOCKS_PER_SEC;
  char* nombreCompletoArchivoTiempo = (char*) malloc(sizeof(char)*strlen(nombreArchivoTiempo)*strlen(nombreDirPrin)+sizeof(char)*3);
  strcpy(nombreCompletoArchivoTiempo, nombreDirPrin);
  strcat(nombreCompletoArchivoTiempo, "/");
  strcat(nombreCompletoArchivoTiempo, nombreArchivoTiempo);
  FILE* archivoTiempo = fopen(nombreCompletoArchivoTiempo, "w");
  float minutitos = time_taken/60;
  float horas = minutitos/60;
  printf("El tiempo de ejecucion fue %.12f segundos o %.12f minutos o %.12f horas.\n", time_taken, minutitos, horas);
  fprintf(archivoTiempo, "El tiempo de ejecucion fue %.12f segundos o %.12f minutos o %.12f horas.\n", time_taken, minutitos, horas);
  fclose(archivoTiempo);

  // // char nombreDirPrin[] = "calCompresiones_Normal";
  // // char nombreArchivoTiempo[] = "tiempo.txt";
  // // int cantParamEvaInfo = 100;
  // // float inicioIntervalo = 1.0;
  // // float finIntervalo = 100.0;
  // // float tolGolden = 1E-12;
  // // float nuevoAncho = 1.0 * delta_u;
  // // clock_t t;
  // // t = clock();
  // // calPSNRDeDistintasCompresiones_Normal(inicioIntervalo, finIntervalo, cantParamEvaInfo, nombreDirPrin, nuevoAncho, maxIter, tolGrad, u, v, w, visi_parteImaginaria, visi_parteReal, delta_u, delta_v, cantVisi, N, matrizDeUnosEstFourier, estrechezDeBorde);
  // // // calPSNRDeDistintasCompresiones_Rect(inicioIntervalo, finIntervalo, cantParamEvaInfo, nombreDirPrin, nuevoAncho, maxIter, tolGrad, u, v, w, visi_parteImaginaria, visi_parteReal, delta_u, delta_v, matrizDeUnos, cantVisi, N, matrizDeUnosEstFourier, estrechezDeBorde);
  // // // calImagenesADistintasCompresiones_Rect(inicioIntervalo, finIntervalo, cantParamEvaInfo, nombreDirPrin, nuevoAncho, maxIter, tolGrad, u, v, w, visi_parteImaginaria, visi_parteReal, delta_u, delta_v, matrizDeUnos, cantVisi, N, matrizDeUnosEstFourier, estrechezDeBorde);
  // // t = clock() - t;
  // // float time_taken = ((float)t)/CLOCKS_PER_SEC;
  // // char* nombreCompletoArchivoTiempo = (char*) malloc(sizeof(char)*strlen(nombreArchivoTiempo)*strlen(nombreDirPrin)+sizeof(char)*3);
  // // strcpy(nombreCompletoArchivoTiempo, nombreDirPrin);
  // // strcat(nombreCompletoArchivoTiempo, "/");
  // // strcat(nombreCompletoArchivoTiempo, nombreArchivoTiempo);
  // // FILE* archivoTiempo = fopen(nombreCompletoArchivoTiempo, "w");
  // // float minutitos = time_taken/60;
  // // float horas = minutitos/60;
  // // printf("El tiempo de ejecucion fue %.12f segundos o %.12f minutos o %.12f horas.\n", time_taken, minutitos, horas);
  // // fprintf(archivoTiempo, "El tiempo de ejecucion fue %.12f segundos o %.12f minutos o %.12f horas.\n", time_taken, minutitos, horas);
  // // fclose(archivoTiempo);
  //
  //
  // // printf("...Comenzando calculo de MV...\n");
  // // clock_t tiempoCalculoMV;
  // // tiempoCalculoMV = clock();
  // // float* MV = calcularMV_Rect(v, delta_v, cantVisi, N, estrechezDeBorde, delta_v, matrizDeUnos);
  // // tiempoCalculoMV = clock() - tiempoCalculoMV;
  // // float tiempoTotalCalculoMV = ((float)tiempoCalculoMV)/CLOCKS_PER_SEC;
  // // printf("Calculo de MV completado.\n");
  // //
  // // printf("...Comenzando calculo de MU...\n");
  // // clock_t tiempoCalculoMU;
  // // tiempoCalculoMU = clock();
  // // float* MU = calcularMV_Rect(u, delta_u, cantVisi, N, estrechezDeBorde, delta_u, matrizDeUnos);
  // // tiempoCalculoMU = clock() - tiempoCalculoMU;
  // // float tiempoTotalCalculoMU = ((float)tiempoCalculoMU)/CLOCKS_PER_SEC;
  // // printf("Calculo de MU completado.\n");
  // //
  // // int blockSize;   // The launch configurator returned block size
  // // int minGridSize; // The minimum grid size needed to achieve the
  // //                  // maximum occupancy for a full device launch
  // // int gridSize;    // The actual grid size needed, based on input size
  // //
  // // hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, transponerMatriz_kernel, 0, 0);
  // // // Round up according to array size
  // // gridSize = (cantVisi*N + blockSize - 1) / blockSize;
  // //
  // // // long cantBloques = ceil((float) cantFilas*N/1024);
  // // // hadamardProduct_kernel<<<gridSize,blockSize>>>(MU, MV, matrizDeUnos, cantVisi, N);
  // // // combinacionLinealMatrices_kernel<<<gridSize,blockSize>>>(5.0, MU, cantVisi, N, 5.0, MV);
  // // transponerMatriz_kernel<<<gridSize,blockSize>>>(MU, matrizDeUnos, cantVisi, N);
  // // hipDeviceSynchronize();
  // //
  // //   // calculate theoretical occupancy
  // // int maxActiveBlocks;
  // // hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, transponerMatriz_kernel, blockSize, 0);
  // //
  // // int device;
  // // hipDeviceProp_t props;
  // // hipGetDevice(&device);
  // // hipGetDeviceProperties(&props, device);
  // //
  // // float occupancy = (maxActiveBlocks * blockSize / props.warpSize) /
  // //                   (float)(props.maxThreadsPerMultiProcessor /
  // //                           props.warpSize);
  // //
  // // printf("Launched blocks of size %d. Theoretical occupancy: %f\n",
  // //        blockSize, occupancy);
  //
  // hipFree(u);
  // hipFree(v);
  // hipFree(w);
  // hipFree(visi_parteImaginaria);
  // hipFree(visi_parteReal);
  // hipFree(matrizDeUnos);
  // hipFree(matrizDeUnosEstFourier);
}
